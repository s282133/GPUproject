#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>


// question, could i pass the shered memory from main? it make sense, where is better to copy N in shared memory?
__constant__ float M_constant[8];

__global__ void memoriaShared(float *M,float *P,float  *N, int Mask_Width,int vector_Width)
{
	extern __shared__ float n_shared[2048];
    int i;


  int tid= blockIdx.x*blockDim.x+threadIdx.x;
  float Pvalue =0;
  int N_start_point = tid - (Mask_Width/2);

  for(i=0; i< 2048; i++ ) {
  		n_shared[i]= N[i];
  	}

  for(int j=0; j< Mask_Width; j++) {

	  if(N_start_point + j >= 0 && N_start_point+j < vector_Width) {

		  if(tid == 1) {

			  printf("aumenti Pvalue con %f, posizione %d \n", n_shared[N_start_point+j],N_start_point+j);
		  }

		  Pvalue += n_shared[N_start_point+j]*M_constant[j];
	  }

  }
  P[tid]= Pvalue;
}

__global__ void powerBigNumbersGPUShared(unsigned char *risultato,unsigned int esponente,unsigned int base ) {

	__shared__ unsigned char risultatoTot[4096];
	__shared__ unsigned char vettoriSupporto[9][4096];
	int passi;
	double risultatoLog;
	int risultatoLogInt;
	int i;
	int j;
	int threadUtili;
	int potenze2;
	int contapotenze;
	int dato1;
	int dato2;
	int risultatoStep;
    unsigned char parteBassaBase;
    unsigned char parteAltaBase;
    int sommo2;
    int sottraggo1;

    unsigned int supporto;
    unsigned int riporto;
    unsigned int riporto16Pari;
    unsigned int riporto16Dispari;
    unsigned int overflow;
    unsigned int overflowPari;
    unsigned int overflowDispari;
    unsigned char supportoParteBassa;
    unsigned char riportoParteBassa;
	// un thread lavora inizialmente su 4 byte, ci sono 1024 thread per block quindi
	// ci servono 4* 1024 per singolo vettore
	// ma noi
	int t = (blockIdx.x*blockDim.x)+threadIdx.x;

	// prima di tutto ogni thread scrive la base dove è giusto scriverla, altrimenti scrive 0
	threadUtili = esponente /2;
	if(esponente % 2 == 1) {
		threadUtili++;
	}
	//debug
	/*if(t== 0){

		printf("thread utili = %d \n",threadUtili);
	} */
	//fine debug

	// la funzionew log non va
	potenze2=2;
	contapotenze=1;
	i=0;
	while(i== 0) {

		if(esponente < potenze2) {
			i=1;
		} else {
			potenze2=potenze2*2;
			contapotenze++;
		}
	}


	//risultatoLog = log2(esponente);
	//risultatoLogInt = log2(esponente);
	/*if(risultatoLog > risultatoLogInt) {

		risultatoLogInt = risultatoLogInt +1;
	} */
	passi=contapotenze ;
	//debug
	/*	if(t== 0){

			printf("pasi utili = %d \n",passi);
		} */
		//fine debug
		parteBassaBase = base;
		base = base >> 8;
		parteAltaBase = base;
		//debug
		if(t== 0){

					printf("parte bassa base = %x \n",parteBassaBase);
					printf("parte alta base = %x \n",parteAltaBase);
	     }
		//fine debug
		// inizializzazione vettore con tutte le basi

		if(t < threadUtili - 1) {
		risultato[t*4] = parteBassaBase;
		risultato[t*4 +1] = parteAltaBase;
		risultato[t*4 +2] = parteBassaBase;
		risultato[t*4 +3] = parteAltaBase;
		} else {

			risultato[t*4] = 0;
		    risultato[t*4 +1] = 0;
			risultato[t*4 +2] = 0;
		    risultato[t*4 +3] = 0;
		}

		if(t == threadUtili - 1) {

			if(esponente % 2 == 1) { // se l'esponente è dispari un thread ha solo mezzo dato
				risultato[t*4] = parteBassaBase;
				risultato[t*4 +1] = parteAltaBase;
				risultato[t*4 +2] = 0;
				risultato[t*4 +3] = 0;

			} else { // se l'esponente è pari è tutto ok

				risultato[t*4] = parteBassaBase;
				risultato[t*4 +1] = parteAltaBase;
				risultato[t*4 +2] = parteBassaBase;
				risultato[t*4 +3] = parteAltaBase;

			}
		}
       __syncthreads();
		//debug
		/*if(t== 0){

			for(i=0;i< 30; i++) {

				printf("risultato[%d] = %d \n",i,risultato[i] );
			}
		} */
				//fine debug


	if(t < threadUtili ) {
	for(i=0; i < passi; i++) {

     if(i== 0){

    	 dato1= risultato[t*4 +1];
    	 dato1 = dato1 << 8;
    	 dato1= dato1 + risultato[t*4];
    	 dato2= risultato[t*4 +3];
    	 dato2 = dato2 << 8;
    	 dato2= dato2 + risultato[t*4 +2];

    	 if(dato2 == 0) {

    		 risultatoStep= dato1;
    		 risultato[t*4] = risultatoStep;
    		 risultatoTot[t*4] = risultatoStep;
    		 risultatoStep = risultatoStep >>8;
    		 risultato[t*4 +1 ] = risultatoStep;
    		 risultatoTot[t*4 +1] = risultatoStep;
    		 risultatoStep = risultatoStep >>8;
    		 risultato[t*4 +2 ] = risultatoStep;
    		 risultatoTot[t*4 +2] = risultatoStep;
    		 risultatoStep = risultatoStep >>8;
    		 risultato[t*4 +3 ] = risultatoStep;
    		 risultatoTot[t*4 +3] = risultatoStep;

    	 } else {

    		 risultatoStep= dato1*dato2;
    		 risultato[t*4] = risultatoStep;
    		 risultatoTot[t*4 ] = risultatoStep;
    		 risultatoStep = risultatoStep >>8;
    		 risultato[t*4 +1 ] = risultatoStep;
    		 risultatoTot[t*4 +1] = risultatoStep;
    		 risultatoStep = risultatoStep >>8;
    		 risultato[t*4 +2 ] = risultatoStep;
    		 risultatoTot[t*4 +2] = risultatoStep;
    		 risultatoStep = risultatoStep >>8;
    		 risultato[t*4 +3 ] = risultatoStep;
    		 risultatoTot[t*4 +3] = risultatoStep;
    	 }

     }
     __syncthreads();
     		//debug
     		if(t== 0){

     			for(i=0;i< 30; i++) {

     				printf("risultato[%d] = %x \n",i,risultato[i] );
     			}
     		}
     				//fine debug
     		/*
     if(i == 1){

    	//  if(t % 2 == 0) {
    	 if(t % 2 == 0) {
    		 sommo2=0;
    		 sottraggo1=0;

    	 } else { // servono a far si che i thread dispari puntino al posto giusto

    		 sommo2=2;
    		 sottraggo1=1;
    	 }
    		  // devo reimpostare base
    		  base = risultatoTot[t*8 +4 +1 + sommo2] ;
              base = base << 8;
              base = base + risultatoTot[t*8 +4 + sommo2] ;
    		  riporto=0;
    		  riporto16Pari=0;
    		  riporto16Dispari=0;
    		  overflow=0;
    		  overflowDispari=0;
    		  overflowPari=0;
    		  for(j=8*(t-sottraggo1); j< 4 + 8*(t-sottraggo1);j++) {

    			  supporto = ((unsigned int) risultatoTot[j]) * base;
    			  supportoParteBassa= supporto;
    			  riportoParteBassa = riporto;

    			  if(j % 2 == 0) {

    			     if( ((unsigned int) supportoParteBassa +  (unsigned int) riportoParteBassa) > 255 ) {
    			         overflowPari=overflowPari +1;
    			      }
    			                          //risultatoInt[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
    			        //risultato[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
    			        vettoriSupporto[t %2][j] = (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
    			     //if( ((unsigned int) risultato[j] +  (unsigned int) riporto16Pari) > 255 ) {
    			      if( ((unsigned int) vettoriSupporto[t %2][j] +  (unsigned int) riporto16Pari) > 255 ) {
    			         overflowPari=overflowPari +1;
    			      }
    			     // risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) riporto16Pari );
    			      vettoriSupporto[t %2][j] = (unsigned char) ( (unsigned int) vettoriSupporto[t %2][j] + (unsigned int) riporto16Pari );

    			     if( ((unsigned int) vettoriSupporto[t %2][j] +  (unsigned int) overflowDispari) > 255 ) {
    			          overflowPari=overflowPari +1;
    			      }
    			     // risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) overflowDispari );
    			      vettoriSupporto[t %2][j] = (unsigned char) ( (unsigned int) vettoriSupporto[t %2][j] + (unsigned int) overflowDispari );
    			      overflowDispari=0;
    			                 // risultato[j]= (char) ( (unsigned int) supporto + (unsigned int) riporto + (unsigned int) riporto16Pari);
    			   } else {

    			     if( ((unsigned int) supportoParteBassa +  (unsigned int) riportoParteBassa) > 255 ) {
    			      overflowDispari=overflowDispari +1;
    			     }
    			     // risultato[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
    			     vettoriSupporto[t %2][j] = (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );

    			     if( ((unsigned int) vettoriSupporto[t %2][j] +  (unsigned int) riporto16Dispari) > 255 ) {
    			      overflowDispari=overflowDispari +1;
    			     }
    			    // risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) riporto16Dispari );
    			     vettoriSupporto[t %2][j] = (unsigned char) ( (unsigned int) vettoriSupporto[t %2][j] + (unsigned int) riporto16Dispari );
    			     if( ((unsigned int) vettoriSupporto[t %2][j] +  (unsigned int) overflowPari) > 255 ) {
    			      overflowDispari=overflowDispari +1;
    			     }
    			     //risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) overflowPari );
    			     vettoriSupporto[t %2][j] = (unsigned char) ( (unsigned int) vettoriSupporto[t %2][j] + (unsigned int) overflowPari );
    			     overflowPari=0;

    			                  //risultato[j]= (char) ( (unsigned int) supporto + (unsigned int) riporto + (unsigned int) riporto16PDispari);
    			    }
    			    supporto= supporto >> 8;

    			                 // supporto = supporto + overflow;
    			                 // overflow=0;
    			    riporto = supporto;
    			    if(j % 2 == 0) {
    			    riporto16Pari = riporto >> 8;
    			    } else {
    			    riporto16Dispari = riporto >> 8;

    			    }
    		  }
    		  overflowPari=0;
    		  riporto16Dispari=0;
    		  // printf("riporto16Pari = %x \n",riporto16Pari);
    		  //printf("supporto = %x \n",supporto);
    		  //printf("overflowDispari = %x \n",overflowDispari);
    		  if( ((unsigned char) supporto  + (unsigned int) riporto16Pari) > 255 ) {
    		     overflowPari=1;
    		               //   printf("overflow viene aumentato 1 \n");
    		  }
    		  //risultato[2 + 2*i] = (unsigned char) supporto  + (unsigned int) riporto16Pari ;
    		  vettoriSupporto[t%2][4 + 8*(t-sottraggo1)] =  (unsigned char) supporto  + (unsigned int) riporto16Pari ;

    		  if( (unsigned int) vettoriSupporto[t %2][4 + 8*(t-sottraggo1)]  + (unsigned int) overflowDispari > 255 ) {
    		      overflowPari = overflowPari+ 1;
    		               // printf("overflow viene aumentato 2 \n");
    		  }
    		  //risultato[2 + 2*i] = (unsigned char) ( (unsigned int) risultato[2 + 2*i] + (unsigned int) overflowDispari );
    		  vettoriSupporto[t%2][4 + 8*(t-sottraggo1)] = (unsigned char) ( (unsigned int) vettoriSupporto[t %2][4 + 8*(t-sottraggo1)]  + (unsigned int) overflowDispari );

    		  supporto= supporto >> 8;
    		  //risultato[2 + 2*i +1] = (unsigned char) supporto  + (unsigned int) overflowPari ;
    		  vettoriSupporto[t%2][4 + 8*(t-sottraggo1) +1] = (unsigned char) supporto  + (unsigned int) overflowPari ;

    		 // }

    		  __syncthreads();

    		  // i thread pari sommano i due risultati per ora in risultato, riciclo overflow pari e overflow dispari
    		  overflowPari = 0;
    		  overflowDispari=0;
    		  if(t % 2 == 0) {  // ATTENZIONE, sostsituire risultato con risultatoTOT

    			  risultato[t*8] = vettoriSupporto[0][t*8];
    			  risultato[t*8 +1] = vettoriSupporto[0][t*8 +1];
    			  if(vettoriSupporto[0][t*8 +2] + vettoriSupporto[1][t*8] > 255) {
    				  overflowPari = 1;
    			  }
    			  risultato[t*8 +2] = vettoriSupporto[0][t*8 +2] + vettoriSupporto[1][t*8];
    			  if(vettoriSupporto[0][t*8 +3] + vettoriSupporto[1][t*8+1 ] + overflowPari > 255) {
    			      overflowDispari = 1;
    			  }
    			  risultato[t*8 +3] = vettoriSupporto[0][t*8 +3] + vettoriSupporto[1][t*8 +1 ] +overflowPari;
    			  if(vettoriSupporto[0][t*8 +4] + vettoriSupporto[1][t*8 +2 ] + overflowDispari  > 255) {
    			      	overflowPari = 1;
    			  }
    			  risultato[t*8 +4] = vettoriSupporto[0][t*8 +4] + vettoriSupporto[1][t*8 +2]+ overflowDispari;
    			  if(vettoriSupporto[0][t*8 +5] + vettoriSupporto[1][t*8+3 ] + overflowPari > 255) {
    			        overflowDispari = 1;
    			  }
    			  risultato[t*8 +5] = vettoriSupporto[0][t*8 +5] + vettoriSupporto[1][t*8 +3 ] +overflowPari;
    			  if(vettoriSupporto[0][t*8 +6] + vettoriSupporto[1][t*8 +4 ] + overflowDispari  > 255) {
    			      	overflowPari = 1;
    			  }
    			  risultato[t*8 +6] = vettoriSupporto[0][t*8 +6] + vettoriSupporto[1][t*8 +4]+ overflowDispari;
    			  if(vettoriSupporto[0][t*8 +7] + vettoriSupporto[1][t*8+5 ] + overflowPari > 255) {
    			      	overflowDispari = 1;
    			  }
    			  risultato[t*8 +5] = vettoriSupporto[0][t*8 +7] + vettoriSupporto[1][t*8 +5 ] +overflowPari;
    		  }
    		  __syncthreads();
    		  if(t== 0) {

    			  for(j=0; j < 7; j++ ) {

    				  printf("risultato[%d] = %d \n",j,risultato[j] );
    				  printf("risultato[%d] = %d \n",j+1024,risultato[j+1024] );
    			  }


    		  }

    	  }
    	 //vettoriSupporto[0][4096]
     */
    /* if(i == 2){


     }
     if(i == 3){


     }

   */
	}


	}
}

__global__ void provacuda(void){


	int tid= blockIdx.x*blockDim.x+threadIdx.x;

	if(tid == 0 ) {

		printf("ciao2 \n");

	}
	printf("ciao \n");
}

__global__ void helloFromGPU(void) {

	printf("hello from GPU \n");
}

int main(void)
{
 // const int n = 2048;
  const int Mask_Width= 8;

 // float *M_device;
 // float *M_host;
 // float *N_device;
 // float *N_host;
 // float *P_device;
 // float *P_host;

  int i;

  unsigned int esponente; // numero massimo su 16 bit, deve essere almeno 2
  unsigned int base;  // numero massimo su 16 bit
  int n =131070;
  unsigned char *dev_vect;
  unsigned char *vect;

  vect = (unsigned char*) malloc(n*sizeof(unsigned char));
  hipMalloc((void**)&dev_vect,n*sizeof(unsigned char));
  hipMemcpy(dev_vect,vect,n*sizeof(unsigned char),hipMemcpyHostToDevice);
  provacuda<<<1,10>>>();
  powerBigNumbersGPUShared<<<32,1024>>>(dev_vect, 4096,7);
  hipDeviceReset();
  return 0;
   //provacuda<<<1,1>>>();
  //powerBigNumbersGPUShared<<<32,1024>>>(dev_vect, 15,7);
   // powerBigNumbersGPUShared<<<1,1024>>>(dev_vect, 15,7);

  //hipMalloc((void**)&var,sizeof(int));
  /*N_host  = (float*) malloc(n*sizeof(float));
  P_host  = (float*) malloc(n*sizeof(float));

  for(i=0; i< n; i++) {
	  P_host[i] = 0;
	  if(i % 2 == 0) {

		  N_host[i] = 0;
	  }else {
		  N_host[i] = 1;
	  }
  }
*/
  /* hipMalloc((void**)&P_device,n*sizeof(float));
  hipMemcpy(P_device,P_host,n*sizeof(float),hipMemcpyHostToDevice);
  hipMalloc((void**)&N_device,n*sizeof(float));
    hipMemcpy(N_device,N_host,n*sizeof(float),hipMemcpyHostToDevice);

  M_host  = (float*) malloc(Mask_Width*sizeof(float));
  */
  /*for(i=0; i< Mask_Width; i++) {

  		  M_host[i] = i;
  		  printf("M [%d] = %f \n",i,M_host[i]);

    } */
  //hipMemcpyToSymbol(HIP_SYMBOL(M_constant),M_host,Mask_Width*sizeof(float));

  //hipMalloc((void**)&M_device,Mask_Width*sizeof(float));
  //hipMemcpy(M_device,M_host,Mask_Width*sizeof(float),hipMemcpyHostToDevice);


  //hipMalloc((void**)&giapassati,384*sizeof(int));
  //vect2 = (double*) malloc(n*sizeof(double));
  //giapassatihost = (int*) malloc(384*sizeof(int));
  //for(i=0; i< 384 ; i++) {

    //	  		giapassatihost[i] = 0;

    //	  	}
/*  hipMemcpy(dev_var_main,var,sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(giapassati,giapassatihost,384*sizeof(int),hipMemcpyHostToDevice);
   hipMalloc((void**)&dev_vect2,n*sizeof(double));
   hipMemcpy(dev_vect2,vect2,n*sizeof(double),hipMemcpyHostToDevice); */
  // hipMalloc(&d_d, n * sizeof(double));
   // run version with static shared memory
   //hipMemcpy(d_d, a, n*sizeof(double), hipMemcpyHostToDevice);


  // memoriaShared<<<2,1024,n*sizeof(float)>>>(M_device,P_device,N_device,Mask_Width,n);
  /* hipMemcpy(P_host, P_device, n*sizeof(float), hipMemcpyDeviceToHost);
   for (int i = 0; i < n; i++) {
	   if(P_host[0] != 12 && i == 0) {

		   printf("errore 2 ! \n" );
		   printf("d[%d] = %f \n",i,P_host[0]);
	   }
	   if(P_host[1] != 10 && i == 1) {

	   		   printf("errore 2 ! \n" );
	   		   printf("d[%d] = %f \n",i,P_host[1]);
	   	   }
	   if(P_host[2] != 15 && i == 2) {

	   		   printf("errore 2 ! \n" );
	   		   printf("d[%d] = %f \n",i,P_host[2]);
	   	   }
	   if(P_host[3] != 12 && i == 3) {

	   	   		   printf("errore 2 ! \n" );
	   	   		   printf("d[%d] = %f \n",i,P_host[3]);
	   	   	   }
	   if(P_host[4] != 16 && i == 4) {

	   	   		   printf("errore 2 ! \n" );
	   	   		   printf("d[%d] = %f \n",i,P_host[4]);
	   	   	   }
	   if(P_host[5] != 12 && i == 5) {

	  	   	   		   printf("errore 2 ! \n" );
	  	   	   		   printf("d[%d] = %f \n",i,P_host[5]);
	  	   	   	   }
	   if(P_host[6] != 16 && i == 6) {

	   	  	   	   		   printf("errore 2 ! \n" );
	   	  	   	   		   printf("d[%d] = %f \n",i,P_host[6]);
	   	  	   	   	   }
	  // printf("d[%i] = %f \n",i,vect2[i]);

   } */
  // run dynamic shared memory version
 // hipMemcpy(d_d, a, n*sizeof(int), hipMemcpyHostToDevice);
 // dynamicReverse<<<1,n,n*sizeof(int)>>>(d_d, n);
 // hipMemcpy(d, d_d, n * sizeof(int), hipMemcpyDeviceToHost);
 // for (int i = 0; i < n; i++)
  //  if (d[i] != r[i]) printf("Error: d[%d]!=r[%d] (%d, %d)n", i, i, d[i], r[i]);


}
