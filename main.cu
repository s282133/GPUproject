
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
//#include <math.h>
/*
__global__ void memoriaGlobale(double *d, int n, int numeroBlocchi)
{
 // __shared__ int s[n];
	int i;

  int t = (blockIdx.x*blockDim.x)+threadIdx.x;
  //printf("t : %d \n",t);
  //int tr = n-t-1;
  //d[t] = t ;
   // printf("t : %d \n", t);
   d[t*16] = t*16 ;
   d[t*16+1] = t*16+1 ;
   d[t*16+2] = t*16+2 ;
   d[t*16+3] = t*16+3 ;
   d[t*16+4] = t*16+4 ;
   d[t*16+5] = t*16+5 ;
   d[t*16+6] = t*16+6 ;
   d[t*16+7] = t*16+7 ;
   d[t*16+8] = t*16+8 ;
   d[t*16+9] = t*16+9 ;
   d[t*16+10] = t*16+10 ;
   d[t*16+11] = t*16+11 ;
   d[t*16+12] = t*16+12 ;
   d[t*16+13] = t*16+13 ;
   d[t*16+14] = t*16+14 ;
   d[t*16+15] = t*16+15 ;
   //printf("d : %f \n", d[t*16+15]);
   printf("error \n");
   for(i=0; i< 16; i++ ) {

	   if(d[t*16+i] != t*16+i) {
		   printf("error \n");
	   }
   }
  // __syncthreads();
  // d[t] = s[tr];
}

__global__ void memoriaShared(double *d, int *giapassati)
{
  //extern __shared__ int s[];
	const int dimMEM = 6144; // perche 6144 è la dimewnsione massima
	const int blocchiDa16 = 100; // 6144/16; // da 16 perche ogni thread lavora su 16 dati
	__shared__ double s[dimMEM];
	//int permessi[blocchiDa16];
	int tArrotondato;
	int i;
	int var=1;
	int contagiapassati=0;
	int controllo=0;


  int t = (blockIdx.x*blockDim.x)+threadIdx.x;
 // if(t < blocchiDa16) {

	//  printf("t generati minori di 100 : %d \n",t);
  //}
  //printf("\n");
  //printf("\n");


   if(t == 97 ) {

  	   printf("ciao sono 97 prima \n ");
     }


   tArrotondato= t % (blocchiDa16);

   if(t == 97 ) {

     	   printf("ciao sono 97 dopo \n tArrotondato = %d \n", tArrotondato);
        }
   //__syncthreads();
  while(t >= blocchiDa16*var) {
	  if(t == 97 ) {

	  	   printf("ciao sono 97 in while \n ");
	     }
  }
   if(t == 97 ) {

	   printf("ciao sono 97 dopo dopo \n ");
   }

  giapassati[tArrotondato] = giapassati[tArrotondato] +1;
  printf("t che è riuscito a passare : %d, contagiapassati : %d \n",tArrotondato, giapassati[tArrotondato]);
   //permessi[tArrotondato] =1;


  s[tArrotondato*16] = t*16;
  s[tArrotondato*16 +1] = t*16 + 1;
  s[tArrotondato*16 +2] = t*16 + 2;
  s[tArrotondato*16 +3] = t*16 + 3;
  s[tArrotondato*16 +4] = t*16 + 4;
  s[tArrotondato*16 +5] = t*16 + 5;
  s[tArrotondato*16 +6] = t*16 + 6;
  s[tArrotondato*16 +7] = t*16 + 7;
  s[tArrotondato*16 +8] = t*16 + 8;
  s[tArrotondato*16 +9] = t*16 + 9;
  s[tArrotondato*16 +10] = t*16 + 10;
  s[tArrotondato*16 +11] = t*16 + 11;
  s[tArrotondato*16 +12] = t*16 + 12;
  s[tArrotondato*16 +13] = t*16 + 13;
  s[tArrotondato*16 +14] = t*16 + 14;
  s[tArrotondato*16 +15] = t*16 + 15;
   // __syncthreads();


  d[t*16] = s[tArrotondato*16];
  d[t*16+1] = s[tArrotondato*16+1];
  d[t*16+2] = s[tArrotondato*16+2];
  d[t*16+3] = s[tArrotondato*16+3];
  d[t*16+4] = s[tArrotondato*16+4];
  d[t*16+5] = s[tArrotondato*16+5];
  d[t*16+6] = s[tArrotondato*16+6];
  d[t*16+7] = s[tArrotondato*16+7];
  d[t*16+8] = s[tArrotondato*16+8];
  d[t*16+9] = s[tArrotondato*16+9];
  d[t*16+10] = s[tArrotondato*16+10];
  d[t*16+11] = s[tArrotondato*16+11];
  d[t*16+12] = s[tArrotondato*16+12];
  d[t*16+13] = s[tArrotondato*16+13];
  d[t*16+14] = s[tArrotondato*16+14];
  d[t*16+15] = s[tArrotondato*16+15];


 // permessi[tArrotondato] =0;
   if(t  == blocchiDa16*var -1) {
       controllo=0;



       while(controllo == 0) {
    	   controllo=1;
    	   for(i=0;i< blocchiDa16; i++) {

    		   if(giapassati[i] ==  0) {
    			   controllo = 0;
    			 //  printf("giapassati[%d] è ancora a zero \n",i);
    		   }
    	   }

       }

	   //var= var+1;
	   printf("sono T  = %d , var = %d, dovrei incrementare var \n",t,var);
   }
}

*/


#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)
__global__ void powerBigNumbersGPUShared(unsigned char *risultato,unsigned char *vettoriSupporto, unsigned int esponente,unsigned int base, int *debug) {

	//__shared__ unsigned char risultatoTot[4096];
	//__shared__ unsigned char vettoriSupporto[9][4096];
	int n =131070;
	int t= blockIdx.x*blockDim.x+threadIdx.x;
	int dato1;
    int dato2;
    int risultatoStep;
    int threadUtili;
    int threadUtili2;
    int potenze2;
    int contapotenze;
    unsigned char parteBassaBase;
    unsigned char parteAltaBase;
    int i;
    int j;
    int k;
    int passi;
    int sommo2;
    int sottraggo1;
    unsigned int supporto;
    unsigned int riporto;
    unsigned int riporto16Pari;
    unsigned int riporto16Dispari;
    unsigned int overflow;
    unsigned int overflowPari;
    unsigned int overflowDispari;
    unsigned char supportoParteBassa;
    unsigned char riportoParteBassa;
    int debuggg;
    int t2;
    t2=t;
   // risultato[0]=0;
    //risultato[1]=0;

   // dato1= risultato[0];
   // dato2 = risultato[1];
   // moltiplicazione= dato1*dato2;

	/*if(t == 0) {

		printf("ciaojsdnk \n");
		printf("moltiplicazione = %d \n",moltiplicazione);
	} */

   // printf("variabili 00 = %d \n",variabili[0]);
   // printf("variabili 00 = %d \n",variabili[1]);
	threadUtili = esponente /2;
		if(esponente % 2 == 1) {
			threadUtili++;
		}
		threadUtili2=threadUtili;
		//debug
		if(t== 0){

			printf("thread utili = %d \n",threadUtili);
		}
		//fine debug

		// la funzionew log non va
		potenze2=2;
		contapotenze=1;
		i=0;
		while(i== 0) {

			if(esponente < potenze2) {
				i=1;
			} else {
				potenze2=potenze2*2;
				contapotenze++;
			}
		}


		//risultatoLog = log2(esponente);
		//risultatoLogInt = log2(esponente);
		/*if(risultatoLog > risultatoLogInt) {

			risultatoLogInt = risultatoLogInt +1;
		} */
		passi=contapotenze ;
		//debug
			if(t== 0){

				//printf("pasi utili = %d \n",passi);
			}
			//fine debug
			parteBassaBase = base;
			base = base >> 8;
			parteAltaBase = base;
			//debug
			if(t== 0){

						//printf("parte bassa base = %x \n",parteBassaBase);
						//printf("parte alta base = %x \n",parteAltaBase);
		     }

			// inizializzazione vettore con tutte le basi

					if(t < threadUtili - 1) {
					risultato[t*4] = parteBassaBase;
					risultato[t*4 +1] = parteAltaBase;
					risultato[t*4 +2] = parteBassaBase;
					risultato[t*4 +3] = parteAltaBase;
					} else {

						risultato[t*4] = 0;
					    risultato[t*4 +1] = 0;
						risultato[t*4 +2] = 0;
					    risultato[t*4 +3] = 0;
					}

					if(t == threadUtili - 1) {

						if(esponente % 2 == 1) { // se l'esponente è dispari un thread ha solo mezzo dato
							risultato[t*4] = parteBassaBase;
							risultato[t*4 +1] = parteAltaBase;
							risultato[t*4 +2] = 0;
							risultato[t*4 +3] = 0;

						} else { // se l'esponente è pari è tutto ok

							risultato[t*4] = parteBassaBase;
							risultato[t*4 +1] = parteAltaBase;
							risultato[t*4 +2] = parteBassaBase;
							risultato[t*4 +3] = parteAltaBase;

						}
					}
			       __syncthreads();
			//fine debug

	   if(t < threadUtili ) {


		for(i=0; i < passi; i++) {

			if(i== 0){

			    	 dato1= risultato[t*4 +1];
			    	 dato1 = dato1 << 8;
			    	 dato1= dato1 + risultato[t*4];
			    	 dato2= risultato[t*4 +3];
			    	 dato2 = dato2 << 8;
			    	 dato2= dato2 + risultato[t*4 +2];
			    	 //dato1 = dato1*dato2; a quanto pare se non riuso dato1 non funziona, e non so il perche
			    	 if(t== 0){
                         printf("threadutili = %d \n", threadUtili2);
			    		 //printf("dato1 = %d \n", dato1);
			    		// printf("dato2 = %d \n", dato2);
			    	 }
			    	/* if(dato1 != 7) {
			    		 printf("err, sono thread %d , dato1 = %d \n ",t,dato1);
			    	 }
			    	 if(dato2 != 7) {
			    	 	 printf("err, sono thread %d , dato2 = %d \n ",t,dato2);
			    	 } */
			    	 __syncthreads();
			    	 if(dato2 == 0) {

			    		 risultatoStep= dato1;
			    		 risultato[t*4] = risultatoStep;
			    		 //risultatoTot[t*4] = risultatoStep;
			    		 risultatoStep = risultatoStep >>8;
			    		 risultato[t*4 +1 ] = risultatoStep;
			    		 //risultatoTot[t*4 +1] = risultatoStep;
			    		 risultatoStep = risultatoStep >>8;
			    		 risultato[t*4 +2 ] = risultatoStep;
			    		// risultatoTot[t*4 +2] = risultatoStep;
			    		 risultatoStep = risultatoStep >>8;
			    		 risultato[t*4 +3 ] = risultatoStep;
			    		 //risultatoTot[t*4 +3] = risultatoStep;

			    	 } else {

			            	// dato1 = ((unsigned int)dato1)+((unsigned int)dato2);
			             risultatoStep= dato1 *dato2;
			    		 risultato[t*4] = risultatoStep;
			    		 //risultatoTot[t*4 ] = risultatoStep;
			    		 risultatoStep = risultatoStep >>8;
                         risultato[t*4 +1 ] = risultatoStep;

			    		 risultatoStep = risultatoStep >>8;

			    		 risultato[t*4 +2 ] = risultatoStep;

			    		 risultatoStep = risultatoStep >>8;

			    		 risultato[t*4 +3 ] = risultatoStep;


			    	 }
			    	 __syncthreads();
                 /* if(t ==0 ) {
                    for(k=0; k< 2048; k++) {

                    	if(k %4 == 0) {
                    		if(risultato[k] != 49) {
                    		    printf("err1, doveva essere 49 [%d] = %d \n ",k,risultato[k]) ;
                    		}

                    	} else {
                    		if(risultato[k] != 0) {
                    			printf("err1, doveva essere 0 [%d] = %d \n ",k,risultato[k]) ;
                    		}
                    	}
                    }
                  }
                  if(t ==2048 ) {
                	  for(k=2048; k< 4096; k++) {

                		  if(k %4 == 0) {
                			  if(risultato[k] != 49) {
                				  printf("err1, doveva essere 49 [%d] = %d \n ",k,risultato[k]) ;
                			  }

                		  } else {
                			  if(risultato[k] != 0) {
                				  printf("err1, doveva essere 0 [%d] = %d \n ",k,risultato[k]) ;
                			  }
                		  }
                	  }
                  } */
			     } // fine i=0

			if(i== 1) {
				if(t < threadUtili ) {
				__syncthreads();
				if(t == 0) {
					for(j=0; j< 30; j++) {
					//printf("risultato[%d] = %d \n",j,risultato[j]);
					}
				}

				 if(t % 2 == 0) {
				    sommo2=0;
				    sottraggo1=0;

				 } else { // servono a far si che i thread dispari puntino al posto giusto
					 sommo2=2;
				     sottraggo1=1;
				 }
				    // devo reimpostare base
				    base = risultato[t*4 +4 +1 + sommo2] ;
				    base = base << 8;
				    base = base + risultato[t*4 +4 + sommo2] ;
				    riporto=0;
				    riporto16Pari=0;
				    riporto16Dispari=0;
				    overflow=0;
				    overflowDispari=0;
				    overflowPari=0;
				    		  if(t== 0) {

				    			 // printf("base = %d \n",base);
				    		  }

				  for(j=4*(t-sottraggo1); j< 4 + 4*(t-sottraggo1);j++) {

				    supporto = ((unsigned int) risultato[j]) * base;
				    supportoParteBassa= supporto;
				    riportoParteBassa = riporto;

				    if(j % 2 == 0) {

				      if( ((unsigned int) supportoParteBassa +  (unsigned int) riportoParteBassa) > 255 ) {
				    	overflowPari=overflowPari +1;
				      }
				       //risultatoInt[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
				    		      			        //risultato[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
				      vettoriSupporto[(t %2)*n + j] = (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
				    		      			     //if( ((unsigned int) risultato[j] +  (unsigned int) riporto16Pari) > 255 ) {
				      if( ((unsigned int) vettoriSupporto[(t %2)*n + j] +  (unsigned int) riporto16Pari) > 255 ) {
				    	overflowPari=overflowPari +1;
				      }
				    		      			     // risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) riporto16Pari );
				      vettoriSupporto[(t %2)*n + j] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %2)*n + j] + (unsigned int) riporto16Pari );

				      if( ((unsigned int) vettoriSupporto[(t %2)*n + j] +  (unsigned int) overflowDispari) > 255 ) {
				        overflowPari=overflowPari +1;
				      }
				    		      			     // risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) overflowDispari );
				      vettoriSupporto[(t %2)*n + j] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %2)*n + j] + (unsigned int) overflowDispari );
				      overflowDispari=0;
				    		      			                 // risultato[j]= (char) ( (unsigned int) supporto + (unsigned int) riporto + (unsigned int) riporto16Pari);
				      } else {

				      if( ((unsigned int) supportoParteBassa +  (unsigned int) riportoParteBassa) > 255 ) {
				       overflowDispari=overflowDispari +1;
				      }
				    		      			     // risultato[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
				      vettoriSupporto[(t %2)*n + j] = (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );

				      if( ((unsigned int) vettoriSupporto[(t %2)*n + j] +  (unsigned int) riporto16Dispari) > 255 ) {
				       overflowDispari=overflowDispari +1;
				      }
				    		      			    // risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) riporto16Dispari );
				      vettoriSupporto[(t %2)*n + j] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %2)*n + j] + (unsigned int) riporto16Dispari );
				      if( ((unsigned int) vettoriSupporto[(t %2)*n + j] +  (unsigned int) overflowPari) > 255 ) {
				       overflowDispari=overflowDispari +1;
				      }
				    		      			     //risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) overflowPari );
				      vettoriSupporto[(t %2)*n + j] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %2)*n + j] + (unsigned int) overflowPari );
				      overflowPari=0;

				    		      			                  //risultato[j]= (char) ( (unsigned int) supporto + (unsigned int) riporto + (unsigned int) riporto16PDispari);
				      }
				      supporto= supporto >> 8;

				    		      			                 // supporto = supporto + overflow;
				    		      			                 // overflow=0;
				      riporto = supporto;
				      if(j % 2 == 0) {
				       riporto16Pari = riporto >> 8;
				      } else {
				       riporto16Dispari = riporto >> 8;

				      }
				    } // chiusura for

				    overflowPari=0;
				    riporto16Dispari=0;
				    		      		  // printf("riporto16Pari = %x \n",riporto16Pari);
				    		      		  //printf("supporto = %x \n",supporto);
				    		      		  //printf("overflowDispari = %x \n",overflowDispari);
				    if( ((unsigned char) supporto  + (unsigned int) riporto16Pari) > 255 ) {
				     overflowPari=1;
				    		      		               //   printf("overflow viene aumentato 1 \n");
				    }
				    		      		  // 8*(t-sottraggo1) serve a fare in modo che ad esempio il thread 2 punti a partire da 8 celle in piu, in modo da saltare thread 0 e 1
				    vettoriSupporto[(t %2)*n + 4 + 4*(t-sottraggo1)] =  (unsigned char) supporto  + (unsigned int) riporto16Pari ;

				    if( (unsigned int) vettoriSupporto[(t %2)*n + 4 + 4*(t-sottraggo1)]  + (unsigned int) overflowDispari > 255 ) {
				     overflowPari = overflowPari+ 1;
				    		      		               // printf("overflow viene aumentato 2 \n");
				    }
				    		      		  //risultato[2 + 2*i] = (unsigned char) ( (unsigned int) risultato[2 + 2*i] + (unsigned int) overflowDispari );
				    vettoriSupporto[(t %2)*n+ 4 + 4*(t-sottraggo1)] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %2)*n+ 4 + 4*(t-sottraggo1)]  + (unsigned int) overflowDispari );

				    supporto= supporto >> 8;
				    		      		  //risultato[2 + 2*i +1] = (unsigned char) supporto  + (unsigned int) overflowPari ;
				    vettoriSupporto[(t %2)*n+ 4 + 4*(t-sottraggo1) +1] = (unsigned char) supporto  + (unsigned int) overflowPari ;

				    		      		 // }

				    __syncthreads();

				    		      		  // i thread pari sommano i due risultati per ora in risultato, riciclo overflow pari e overflow dispari
				    overflowPari = 0;
				    overflowDispari=0;
				    if(t % 2 == 0) {  // ATTENZIONE, sostsituire risultato con risultatoTOT

				     risultato[t*4] = vettoriSupporto[t*4];
				     /*if(t*4 +2== 2042) {
				    	 printf("scrivo su 2042 = %d \n",risultato[t*4]);
				     }
                      */
				     risultato[t*4 +1] = vettoriSupporto[t*4 +1];
				    if(vettoriSupporto[t*4 +2] + vettoriSupporto[n+ t*4] > 255) {
				     overflowPari = 1;
				    }
				    risultato[t*4 +2] = vettoriSupporto[t*4 +2] + vettoriSupporto[n+ t*4];
				   /* if(t == 510) {
				    	printf("vettoriSupporto[t*4 +2] = %d \n",vettoriSupporto[t*4 +2]);
				    	printf("vettoriSupporto[n+ t*4] = %d \n",vettoriSupporto[n+ t*4]);
				    } */
				    if(vettoriSupporto[t*4 +3] + vettoriSupporto[n+ t*4+1 ] + overflowPari > 255) {
				     overflowDispari = 1;
				    }
				    risultato[t*4 +3] = vettoriSupporto[t*4 +3] + vettoriSupporto[n + t*4 +1 ] +overflowPari;
				    if(vettoriSupporto[t*4 +4] + vettoriSupporto[n + t*4 +2 ] + overflowDispari  > 255) {
				     overflowPari = 1;
				    }
				    risultato[t*4 +4] = vettoriSupporto[t*4 +4] + vettoriSupporto[n+ t*4 +2]+ overflowDispari;
				    if(vettoriSupporto[t*4 +5] + vettoriSupporto[n + t*4+3 ] + overflowPari > 255) {
				     overflowDispari = 1;
				    }
				    risultato[t*4 +5] = vettoriSupporto[t*4 +5] + vettoriSupporto[n + t*4 +3 ] +overflowPari;
				    if(vettoriSupporto[t*4 +6] + vettoriSupporto[n + t*4 +4 ] + overflowDispari  > 255) {
				     overflowPari = 1;
				    }
				    risultato[t*4 +6] =  vettoriSupporto[n + t*4 +4]+ overflowDispari; // se ci sono problemi controllare qui
				    risultato[t*4 +7] =  vettoriSupporto[n + t*4 +5 ] +overflowPari;
				   }

				    __syncthreads();
				    // debug
				  /*  if(t == 0) { // i thread sono 2047
                    //printf("ciao asdmkfnm \n");
				     for(k=0; k < 2048; k++ ) {

				    	 if(k %8 == 0) {
				    		 if(risultato[k] != 97) {
				    			 printf("err doveva essere 97, risultato[%d]= %d \n ",k,risultato[k]);
				    		 }
				    	 }
				    	 if(k %8 == 1) {
				    		 if(risultato[k] != 9) {
				    		 	printf("err doveva essere 9,risultato[%d]= %d \n ",k,risultato[k]);
				    		 }

				    	  } else {
				    		  if(risultato[k] != 0 && k %8 != 0) {
				    		  	printf("err doveva essere 0, risultato[%d]= %d \n ",k,risultato[k]);
				    		  }
				    	  }
				      //printf(" hghj risultato[%d] = %x \n",k,risultato[k] );
				     // printf("ghv risultato[%d] = %d \n",k+1024,risultato[k+1024] );
				     }


				    }
				    __syncthreads();
				    if(t == 512) { // i thread sono 2047
				    	//printf("ciao asdmkfnm \n");
				    	for(k=2048; k < 2048; k++ ) {

				    		if(k %8 == 0) {
				    			if(risultato[k] != 97) {
				    				printf("err doveva essere 97, risultato[%d]= %d \n ",k,risultato[k]);
				    			}
				    		}
				    		if(k %8 == 1) {
				    			if(risultato[k] != 9) {
				    				printf("err doveva essere 9,risultato[%d]= %d \n ",k,risultato[k]);
				    			}

				    		} else {
				    			if(risultato[k] != 0 && k %8 != 0) {
				    				printf("err doveva essere 0, risultato[%d]= %d \n ",k,risultato[k]);
				    			}
				    		}
				    		//printf(" hghj risultato[%d] = %x \n",k,risultato[k] );
				    		// printf("ghv risultato[%d] = %d \n",k+1024,risultato[k+1024] );
				    	}


				    } */
				    // debug

				} // chiusura thread utili
			} // fine i=1, fino a qui va

		}
	   }

return;
}

__global__ void powerBigNumbersGPUStep2(unsigned char *risultato,unsigned char *vettoriSupporto, unsigned int esponente,unsigned int base, int *debug) {

	    int n =131070;
		int t= blockIdx.x*blockDim.x+threadIdx.x;
		int t2;
		int dato1;
	    int dato2;
	    int risultatoStep;
	    int threadUtili;
	    int potenze2;
	    int contapotenze;
	    unsigned char parteBassaBase;
	    unsigned char parteAltaBase;
	    int i;
	    int j;
	    int k;
	    int passi;
	    int sommo2;
	    int sottraggo1;
	    unsigned int supporto;
	    unsigned int riporto;
	    unsigned int riporto16Pari;
	    unsigned int riporto16Dispari;
	    unsigned int overflow;
	    unsigned int overflowPari;
	    unsigned int overflowDispari;
	    unsigned char supportoParteBassa;
	    unsigned char riportoParteBassa;

	    threadUtili=2048;
		//vettoriSupporto[0] = 0;
		if(t < threadUtili ) {


			//debug[0]=0;
			//__syncthreads();
			//debug[0]= debug[0] +1;


			__syncthreads();


			if(t % 4 == 0) {
				sommo2=0;
				sottraggo1=0;
			}
			if(t % 4 == 1) {
				sommo2=2;
				sottraggo1=1;
			}
			if(t % 4 == 2) {
				sommo2=4;
				sottraggo1=2;
			}
			if(t % 4 == 3) {
				sommo2=6;
				sottraggo1=3;
			}


			base=0;

			//printf("ciao = sono 2047 e sto accedendo a = %d \n",risultato[8176]);
			// debuggg= t*4;
			//base=t2 ;
			t2= (t - (t%4))*4 +8 +1 + (t % 4)*2;
			//base=base+8 +1 + sommo2;
			//printf("t2 = %d \n",t2);
			//printf("ovvero = %d \n",risultato[(t - (t%4))*4 + 8 + 1 + (t % 4)*2]);

			//printf("risultato = %d \n",risultato[t*8 +8 +1 + sommo2]);// devo reimpostare base
			t2=(t-sottraggo1)*4 +8 +1 + sommo2;
			// base= risultato[(t - (t%4))*4 + 8 + 1 + (t % 4)*2] ;
			base = risultato[(t-sottraggo1)*4 +8 +1 + sommo2] ;
			//printf("base = %d \n",base);
			base = base << 8;
			t2=(t-sottraggo1)*4 +8 + sommo2;
			//  base= base + risultato[(t - (t%4))*4 + 8 + (t % 4)*2];
			base = base + risultato[(t-sottraggo1)*4 +8 + sommo2] ;
			// printf("base = %d \n",base);
			/*if(t== 512) {

				printf("base 512= %d \n", base);
			} */
			if(base == 2401) {
				//printf("base corretta, T = %d \n", t);
			} else {
				//printf("base errata, T = %d \n", t);
			}
			riporto=0;
			riporto16Pari=0;
			riporto16Dispari=0;
			overflow=0;
			overflowDispari=0;
			overflowPari=0;
			//debug


			//	if(t== 2047) {
			//vettoriSupporto[(t %4)*n + k + 4*(t-sottraggo1) +1];
			//  printf("base = %d \n",base);
			//}
			//debug

			for(j=4*(t-sottraggo1); j< 8 + 4*(t-sottraggo1);j++) {

				supporto = ((unsigned int) risultato[j]) * base;
				supportoParteBassa= supporto;
				riportoParteBassa = riporto;

				if(j % 2 == 0) {

					if( ((unsigned int) supportoParteBassa +  (unsigned int) riportoParteBassa) > 255 ) {
						overflowPari=overflowPari +1;
					}
					//risultatoInt[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
					//risultato[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
					vettoriSupporto[(t %4)*n + j ] = (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
					//if( ((unsigned int) risultato[j] +  (unsigned int) riporto16Pari) > 255 ) {
					if( ((unsigned int) vettoriSupporto[(t %4)*n + j ] +  (unsigned int) riporto16Pari) > 255 ) {
						overflowPari=overflowPari +1;
					}
					// risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) riporto16Pari );
					vettoriSupporto[(t %4)*n + j ] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %4)*n + j ] + (unsigned int) riporto16Pari );

					if( ((unsigned int) vettoriSupporto[(t %4)*n + j ] +  (unsigned int) overflowDispari) > 255 ) {
						overflowPari=overflowPari +1;
					}
					// risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) overflowDispari );
					vettoriSupporto[(t %4)*n + j ] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %4)*n + j ] + (unsigned int) overflowDispari );
					overflowDispari=0;
					// risultato[j]= (char) ( (unsigned int) supporto + (unsigned int) riporto + (unsigned int) riporto16Pari);
				} else {

					if( ((unsigned int) supportoParteBassa +  (unsigned int) riportoParteBassa) > 255 ) {
						overflowDispari=overflowDispari +1;
					}
					// risultato[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
					vettoriSupporto[(t %4)*n + j ] = (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );

					if( ((unsigned int) vettoriSupporto[(t %4)*n + j ] +  (unsigned int) riporto16Dispari) > 255 ) {
						overflowDispari=overflowDispari +1;
					}
					// risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) riporto16Dispari );
					vettoriSupporto[(t %4)*n + j] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %4)*n + j] + (unsigned int) riporto16Dispari );
					if( ((unsigned int) vettoriSupporto[(t %4)*n + j ] +  (unsigned int) overflowPari) > 255 ) {
						overflowDispari=overflowDispari +1;
					}
					//risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) overflowPari );
					vettoriSupporto[(t %4)*n + j ] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %4)*n + j ] + (unsigned int) overflowPari );
					overflowPari=0;

					//risultato[j]= (char) ( (unsigned int) supporto + (unsigned int) riporto + (unsigned int) riporto16PDispari);
				}
				supporto= supporto >> 8;

				// supporto = supporto + overflow;
				// overflow=0;
				riporto = supporto;
				if(j % 2 == 0) {
					riporto16Pari = riporto >> 8;
				} else {
					riporto16Dispari = riporto >> 8;

				}
			} // chiusura for

			overflowPari=0;
			riporto16Dispari=0;
			// printf("riporto16Pari = %x \n",riporto16Pari);
			//printf("supporto = %x \n",supporto);
			//printf("overflowDispari = %x \n",overflowDispari);
			if( ((unsigned char) supporto  + (unsigned int) riporto16Pari) > 255 ) {
				overflowPari=1;
				//   printf("overflow viene aumentato 1 \n");
			}
			//risultato[2 + 2*i] = (unsigned char) supporto  + (unsigned int) riporto16Pari ;
			vettoriSupporto[(t %4)*n + 8 + 4*(t-sottraggo1)] =  (unsigned char) supporto  + (unsigned int) riporto16Pari ;

			if( (unsigned int) vettoriSupporto[(t %4)*n + 8 + 4*(t-sottraggo1)]  + (unsigned int) overflowDispari > 255 ) {
				overflowPari = overflowPari+ 1;
				// printf("overflow viene aumentato 2 \n");
			}
			//risultato[2 + 2*i] = (unsigned char) ( (unsigned int) risultato[2 + 2*i] + (unsigned int) overflowDispari );
			vettoriSupporto[(t %4)*n+ 8+ 4*(t-sottraggo1)] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %4)*n+ 8 + 4*(t-sottraggo1)]  + (unsigned int) overflowDispari );

			supporto= supporto >> 8;
			//risultato[2 + 2*i +1] = (unsigned char) supporto  + (unsigned int) overflowPari ;
			vettoriSupporto[(t %4)*n+ 8 + 4*(t-sottraggo1) +1] = (unsigned char) supporto  + (unsigned int) overflowPari ;

			// }

			if(t == 512) {

				//printf(" 512, vettoriSupporto[2048]= %d \n", vettoriSupporto[512*4]);
				//printf(" 512, vettoriSupporto[2049]= %d \n", vettoriSupporto[512*4 +1 ]);
				//printf(" 512, vettoriSupporto[2050]= %d \n", vettoriSupporto[512*4 +2 ]);
			}
			__syncthreads();

			// i thread pari sommano i due risultati per ora in risultato, riciclo overflow pari e overflow dispari
			overflowPari = 0;
			overflowDispari=0;
			if(t % 4 == 0 || t %4 ==2) { // ATTENZIONE, sostsituire risultato con risultatoTOT

				//offsetT1= t*2;
				//offsetT2= offsetT1 +2;
				for(k=0; k< 12; k++) {

					if(k== 0 || k== 1) {
						//vettoriSupporto[(t %4)*n + k] = vettoriSupporto[(t %4)*n + k];
					}
					if( k> 1 && k < 10) {

						vettoriSupporto[(t %4)*n + k + 4*(t-sottraggo1) +1]= vettoriSupporto[(t %4)*n + k + 4*(t-sottraggo1) +1] + vettoriSupporto[((t+1) %4)*n + k -2 + 4*(t-sottraggo1) +1] +  overflowPari ;
						if(vettoriSupporto[(t %4)*n + k + 4*(t-sottraggo1) +1] + vettoriSupporto[((t+1) %4)*n + k -2 + 4*(t-sottraggo1) +1] + overflowPari > 255) {
							overflowPari = 1;
						} else {
							overflowPari = 0;
						}

					}
					if( k== 10 || k== 11) {
						vettoriSupporto[(t %4)*n + k + 4*(t-sottraggo1) +1]=  vettoriSupporto[((t+1) %4)*n + k -2 + 4*(t-sottraggo1) +1] +  overflowPari ;
						if( vettoriSupporto[((t+1) %4)*n + k -2 + 4*(t-sottraggo1) +1] + overflowPari > 255) {
							overflowPari = 1;
						} else {
							overflowPari = 0;
						}

					}

				}  // fine somme turno 1
				__syncthreads();
				if(t == 512) {
					//printf(" 512, vettoriSupporto[2048] dopo somme= %d \n", vettoriSupporto[512*4]);
					//printf(" 512, vettoriSupporto[2049] dopo somme= %d \n", vettoriSupporto[512*4 +1 ]);
					//printf(" 512, vettoriSupporto[2050] dopo somme= %d \n", vettoriSupporto[512*4 +2 ]);
				}
				overflowPari = 0;
				if(t % 4 == 0 ) {

					for(k=0; k< 16; k++) {

						if(k== 0 || k== 1 || k== 2 || k== 3) {
							//vettoriSupporto[(t %4)*n + k] = vettoriSupporto[(t %4)*n + k];
							risultato[t*4 + k] = vettoriSupporto[(t %4)*n + k];
							// printf(" vettoriSupporto[%d] = %x \n",k,vettoriSupporto[(t %4)*n + k] );
						}
						if( k> 3 && k < 12) {

							risultato[t*4 + k] = vettoriSupporto[(t %4)*n + k + 4*(t-sottraggo1) +1] + vettoriSupporto[((t+2) %4)*n + k -4 + 4*(t-sottraggo1) +1] +  overflowPari ;
							//vettoriSupporto[(t %4)*n + k + 4*(t-sottraggo1) +1]= vettoriSupporto[(t %4)*n + k + 4*(t-sottraggo1) +1] + vettoriSupporto[((t+1) %4)*n + k -2 + 4*(t-sottraggo1) +1] +  overflowPari ;
							if(vettoriSupporto[(t %4)*n + k + 4*(t-sottraggo1) +1] + vettoriSupporto[((t+2) %4)*n + k -4 + 4*(t-sottraggo1) +1] + overflowPari > 255) {
								overflowPari = 1;
							} else {
								overflowPari = 0;
							}

						}
						if( k >= 12) {
							risultato[t*4 + k] = vettoriSupporto[((t+2) %4)*n + k -4 + 4*(t-sottraggo1) +1] +  overflowPari ;
							vettoriSupporto[(t %4)*n + k + 4*(t-sottraggo1) +1]=  vettoriSupporto[((t+2) %4)*n + k -4 + 4*(t-sottraggo1) +1] +  overflowPari ;
							if( vettoriSupporto[((t+2) %4)*n + k -4 + 4*(t-sottraggo1) +1] + overflowPari > 255) {
								overflowPari = 1;
							} else {
								overflowPari = 0;
							}

						}


					}  // fine somme turno 2

					if(t== 512) {

						//printf(" 512, risultato[2048]= %d \n", risultato[2048]);
					}

				}
			}
			__syncthreads();

			//
		} // chiusura threadutili2


	return;
}

__global__ void powerBigNumbersGPUStep3(unsigned char *risultato,unsigned char *vettoriSupporto, unsigned int esponente,unsigned int base, int *debug) {

	int n =131070;
	int t= blockIdx.x*blockDim.x+threadIdx.x;
	int t2;
	int dato1;
	int dato2;
	int risultatoStep;
	int threadUtili;
	int potenze2;
	int contapotenze;
	unsigned char parteBassaBase;
	unsigned char parteAltaBase;
	int i;
	int j;
	int k;
	int passi;
	int sommo2;
	int sottraggo1;
	unsigned int supporto;
	unsigned int riporto;
	unsigned int riporto16Pari;
	unsigned int riporto16Dispari;
	unsigned int overflow;
	unsigned int overflowPari;
	unsigned int overflowDispari;
	unsigned char supportoParteBassa;
	unsigned char riportoParteBassa;

	threadUtili=2048;

	sommo2=t%8*2;
	sottraggo1=t%8;

	base = risultato[(t-sottraggo1)*4 +16 +1 + sommo2] ;
	base = base << 8;
	base = base + risultato[(t-sottraggo1)*4 +16 + sommo2] ;
	riporto=0;
	riporto16Pari=0;
	riporto16Dispari=0;
	overflow=0;
	overflowDispari=0;
	overflowPari=0;
	/*if(t== 544) {
		printf("base = %x \n",base);
	} // fino a qui va */
	/*if(t== 545) {
			printf("base = %x \n",base);
	} // fino a qui va */
	for(j=4*(t-sottraggo1); j< 16 + 4*(t-sottraggo1);j++) {

		/*if(t== 544) {
				printf("j = %d \n",j);
	    } // fino a qui va */

		supporto = ((unsigned int) risultato[j]) * base;
		/*if(t==0) {
			printf("supporto= %d \n",supporto);
		} */
		supportoParteBassa= supporto;
		riportoParteBassa = riporto;

		if(j % 2 == 0) {

			if( ((unsigned int) supportoParteBassa +  (unsigned int) riportoParteBassa) > 255 ) {
				overflowPari=overflowPari +1;
			}
			//risultatoInt[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
			//risultato[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
			vettoriSupporto[(t %8)*n + j ] = (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
			//if( ((unsigned int) risultato[j] +  (unsigned int) riporto16Pari) > 255 ) {
			if( ((unsigned int) vettoriSupporto[(t %8)*n + j ] +  (unsigned int) riporto16Pari) > 255 ) {
				overflowPari=overflowPari +1;
			}
			// risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) riporto16Pari );
			vettoriSupporto[(t %8)*n + j ] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %8)*n + j ] + (unsigned int) riporto16Pari );

			if( ((unsigned int) vettoriSupporto[(t %8)*n + j ] +  (unsigned int) overflowDispari) > 255 ) {
				overflowPari=overflowPari +1;
			}
			// risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) overflowDispari );
			vettoriSupporto[(t %8)*n + j ] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %8)*n + j ] + (unsigned int) overflowDispari );
			overflowDispari=0;
			// risultato[j]= (char) ( (unsigned int) supporto + (unsigned int) riporto + (unsigned int) riporto16Pari);
		} else {

			if( ((unsigned int) supportoParteBassa +  (unsigned int) riportoParteBassa) > 255 ) {
				overflowDispari=overflowDispari +1;
			}
			// risultato[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
			vettoriSupporto[(t %8)*n + j ] = (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );

			if( ((unsigned int) vettoriSupporto[(t %8)*n + j ] +  (unsigned int) riporto16Dispari) > 255 ) {
				overflowDispari=overflowDispari +1;
			}
			// risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) riporto16Dispari );
			vettoriSupporto[(t %8)*n + j] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %8)*n + j] + (unsigned int) riporto16Dispari );
			if( ((unsigned int) vettoriSupporto[(t %8)*n + j ] +  (unsigned int) overflowPari) > 255 ) {
				overflowDispari=overflowDispari +1;
			}
			//risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) overflowPari );
			vettoriSupporto[(t %8)*n + j ] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %8)*n + j ] + (unsigned int) overflowPari );
			overflowPari=0;

			//risultato[j]= (char) ( (unsigned int) supporto + (unsigned int) riporto + (unsigned int) riporto16PDispari);
		}
		supporto= supporto >> 8;

		// supporto = supporto + overflow;
		// overflow=0;
		riporto = supporto;
		if(j % 2 == 0) {
			riporto16Pari = riporto >> 8;
		} else {
			riporto16Dispari = riporto >> 8;

		}
	} // chiusura for

	overflowPari=0;
	riporto16Dispari=0;
	// printf("riporto16Pari = %x \n",riporto16Pari);
	//printf("supporto = %x \n",supporto);
	//printf("overflowDispari = %x \n",overflowDispari);
	if( ((unsigned char) supporto  + (unsigned int) riporto16Pari) > 255 ) {
		overflowPari=1;
		//   printf("overflow viene aumentato 1 \n");
	}
	//risultato[2 + 2*i] = (unsigned char) supporto  + (unsigned int) riporto16Pari ;
	vettoriSupporto[(t %8)*n + 16 + 4*(t-sottraggo1)] =  (unsigned char) supporto  + (unsigned int) riporto16Pari ;

	if( (unsigned int) vettoriSupporto[(t %8)*n + 16 + 4*(t-sottraggo1)]  + (unsigned int) overflowDispari > 255 ) {
		overflowPari = overflowPari+ 1;
		// printf("overflow viene aumentato 2 \n");
	}
	//risultato[2 + 2*i] = (unsigned char) ( (unsigned int) risultato[2 + 2*i] + (unsigned int) overflowDispari );
	vettoriSupporto[(t %8)*n+ 16+ 4*(t-sottraggo1)] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %8)*n+ 16 + 4*(t-sottraggo1)]  + (unsigned int) overflowDispari );

	supporto= supporto >> 8;
	//risultato[2 + 2*i +1] = (unsigned char) supporto  + (unsigned int) overflowPari ;
	/* if(t==0) {

	    	printf("vettoriSupporto[18] prima delle somme= %x \n",vettoriSupporto[18]);
	    } */
	vettoriSupporto[(t %8)*n+ 16 + 4*(t-sottraggo1) +1] = (unsigned char) supporto  + (unsigned int) overflowPari ;

	// }

	/*if(t==512) {
				printf("vettoriSupporto[0] 512= %x \n",vettoriSupporto[2048]);
				printf("vettoriSupporto[1] 512= %x \n",vettoriSupporto[2049]);
				printf("vettoriSupporto[2] 512= %x \n",vettoriSupporto[2050]);
				printf("vettoriSupporto[3] 512= %x \n",vettoriSupporto[2051]);
				printf("vettoriSupporto[4] 512= %x \n",vettoriSupporto[2052]);
				printf("vettoriSupporto[5] 512= %x \n",vettoriSupporto[2053]);
				//printf("vettoriSupporto[18] prima delle somme= %x \n",vettoriSupporto[18]);
			}
	if(t==513) {
					printf("vettoriSupporto[0 +n] 512= %x \n",vettoriSupporto[2048 + (t %8)*n]);
					printf("vettoriSupporto[1 +n] 512= %x \n",vettoriSupporto[2049 + (t %8)*n]);
					printf("vettoriSupporto[2 +n] 512= %x \n",vettoriSupporto[2050 + (t %8)*n]);
					printf("vettoriSupporto[3 +n] 512= %x \n",vettoriSupporto[2051 + (t %8)*n]);
					printf("vettoriSupporto[4 +n] 512= %x \n",vettoriSupporto[2052 + (t %8)*n]);
					printf("vettoriSupporto[5 +n] 512= %x \n",vettoriSupporto[2053 + (t %8)*n]);
					//printf("vettoriSupporto[18] prima delle somme= %x \n",vettoriSupporto[18]);
				} */

	/*if(t==544) {
					printf("vettoriSupporto[0] 544= %x \n",vettoriSupporto[2176]);
					printf("vettoriSupporto[1] 544= %x \n",vettoriSupporto[2177]);
					printf("vettoriSupporto[2] 544= %x \n",vettoriSupporto[2178]);
					printf("vettoriSupporto[3] 544= %x \n",vettoriSupporto[2179]);
					printf("vettoriSupporto[4] 544= %x \n",vettoriSupporto[2180]);
					printf("vettoriSupporto[5] 544= %x \n",vettoriSupporto[2181]);
					printf("vettoriSupporto[6] 544= %x \n",vettoriSupporto[2182]);
					printf("vettoriSupporto[7] 544= %x \n",vettoriSupporto[2183]);
					printf("vettoriSupporto[8] 544= %x \n",vettoriSupporto[2184]);
					printf("vettoriSupporto[9] 544= %x \n",vettoriSupporto[2184]);
					//printf("vettoriSupporto[18] prima delle somme= %x \n",vettoriSupporto[18]);
				}
		if(t==545) {
						printf("vettoriSupporto[0 +n] 545= %x \n",vettoriSupporto[2176 + (t %8)*n]);
						printf("vettoriSupporto[1 +n] 545= %x \n",vettoriSupporto[2177 + (t %8)*n]);
						printf("vettoriSupporto[2 +n] 545= %x \n",vettoriSupporto[2178 + (t %8)*n]);
						printf("vettoriSupporto[3 +n] 545= %x \n",vettoriSupporto[2179 + (t %8)*n]);
						printf("vettoriSupporto[4 +n] 545= %x \n",vettoriSupporto[2180 + (t %8)*n]);
						printf("vettoriSupporto[5 +n] 545= %x \n",vettoriSupporto[2181 + (t %8)*n]);
						//printf("vettoriSupporto[18] prima delle somme= %x \n",vettoriSupporto[18]);
					} */


	/*if(t==0) {
			printf("vettoriSupporto[0] = %x \n",vettoriSupporto[0]);
			printf("vettoriSupporto[1] = %x \n",vettoriSupporto[1]);
			printf("vettoriSupporto[2] = %x \n",vettoriSupporto[2]);
			printf("vettoriSupporto[3] = %x \n",vettoriSupporto[3]);
			printf("vettoriSupporto[4] = %x \n",vettoriSupporto[4]);
			printf("vettoriSupporto[4] = %x \n",vettoriSupporto[5]);
			printf("vettoriSupporto[18] prima delle somme= %x \n",vettoriSupporto[18]);
		}
		if(t==1) {
				printf("vettoriSupporto[0] 2= %x \n",vettoriSupporto[(t %8)*n + 0]);
				printf("vettoriSupporto[1] 2= %x \n",vettoriSupporto[(t %8)*n + 1]);
				printf("vettoriSupporto[2] 2= %x \n",vettoriSupporto[(t %8)*n + 2]);
				printf("vettoriSupporto[3] 2= %x \n",vettoriSupporto[ (t %8)*n + 3]);
				printf("vettoriSupporto[4] 2= %x \n",vettoriSupporto[(t %8)*n + 4]);
				printf("vettoriSupporto[5] 2= %x \n",vettoriSupporto[(t %8)*n + 5]);
			}
		if(t==2) {
					printf("vettoriSupporto[0] 3= %x \n",vettoriSupporto[(t %8)*n + 0]);
					printf("vettoriSupporto[1] 3= %x \n",vettoriSupporto[(t %8)*n + 1]);
					printf("vettoriSupporto[2] 3= %x \n",vettoriSupporto[(t %8)*n + 2]);
					printf("vettoriSupporto[3] 3= %x \n",vettoriSupporto[ (t %8)*n + 3]);
					printf("vettoriSupporto[4] 3= %x \n",vettoriSupporto[(t %8)*n + 4]);
					printf("vettoriSupporto[5] 3= %x \n",vettoriSupporto[(t %8)*n + 5]);
				} */
	__syncthreads();
	// i thread pari sommano i due risultati per ora in risultato, riciclo overflow pari e overflow dispari
	overflowPari = 0;
	overflowDispari=0;
	if(t % 8 == 0 || t %8  ==2 || t % 8 == 4 || t % 8 == 6) { // ATTENZIONE, sostsituire risultato con risultatoTOT

		//offsetT1= t*2;
		//offsetT2= offsetT1 +2;
		for(k=0; k< 20; k++) {

			if(k== 0 || k== 1) {
				//vettoriSupporto[(t %4)*n + k] = vettoriSupporto[(t %4)*n + k];
			}
			if( k> 1 && k < 18) {

				/*if(k== 2 && t==512) {

					printf("sto per sommare: vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) +1]= %x \n",vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) ]);
					printf("sto per sommare: vettoriSupporto[((t+1) %8)*n + k -2 + 4*(t-sottraggo1) +1]= %x \n",vettoriSupporto[((t+1) %8)*n + k -2 + 4*(t-sottraggo1) ]);
				} */

				//vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) ]= vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) ] + vettoriSupporto[((t+1) %8)*n + k -2 + 4*(t-sottraggo1) ] +  overflowPari ;
				if(vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) ] + vettoriSupporto[((t+1) %8)*n + k -2 + 4*(t-sottraggo1) ] + overflowPari > 255) {
					//overflowPari = 1;
					overflowDispari=1;
				} else {
					//overflowPari = 0;
					overflowDispari=0;
				}
				vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) ]= vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) ] + vettoriSupporto[((t+1) %8)*n + k -2 + 4*(t-sottraggo1) ] +  overflowPari ;
                if(overflowDispari == 1) {
                	overflowPari = 1;
                } else {
                	overflowPari = 0;
                }
			}
			if( k== 18 || k== 19) {

				if( vettoriSupporto[((t+1) %8)*n + k -2 + 4*(t-sottraggo1) ] + overflowPari > 255) {
									//overflowPari = 1;
									overflowDispari=1;
								} else {
									//overflowPari = 0;
									overflowDispari=0;
								}
				vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) ]=  vettoriSupporto[((t+1) %8)*n + k -2 + 4*(t-sottraggo1) ] +  overflowPari ;
								if(overflowDispari == 1) {
									overflowPari = 1;
								} else {
									overflowPari = 0;
								}

				/*vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) ]=  vettoriSupporto[((t+1) %8)*n + k -2 + 4*(t-sottraggo1) ] +  overflowPari ;
				if( vettoriSupporto[((t+1) %8)*n + k -2 + 4*(t-sottraggo1) ] + overflowPari > 255) {
					overflowPari = 1;
				} else {
					overflowPari = 0;
				} */

			}

		}
	}// fine somme turno 1
	__syncthreads();
	/*if( t== 0) {

		printf("vettoriSupporto[0] = %x \n",vettoriSupporto[0]);
		printf("vettoriSupporto[1] = %x \n",vettoriSupporto[1]);
		printf("vettoriSupporto[2] = %x \n",vettoriSupporto[2]);
		printf("vettoriSupporto[3] = %x \n",vettoriSupporto[3]);
		printf("vettoriSupporto[4] = %x \n",vettoriSupporto[4]);
		printf("vettoriSupporto[4] = %x \n",vettoriSupporto[5]);
		printf("vettoriSupporto[17] = %x \n",vettoriSupporto[17]);
		printf("vettoriSupporto[18] turno 2= %x \n",vettoriSupporto[18]);
	} */
	/*if(t==512) {
					printf("vettoriSupporto[0] turno 1 512= %x \n",vettoriSupporto[2048]);
					printf("vettoriSupporto[1] turno 1 512= %x \n",vettoriSupporto[2049]);
					printf("vettoriSupporto[2] turno 1 512= %x \n",vettoriSupporto[2050]);
					printf("vettoriSupporto[3] turno 1 512= %x \n",vettoriSupporto[2051]);
					printf("vettoriSupporto[4] turno 1 512= %x \n",vettoriSupporto[2052]);
					printf("vettoriSupporto[5] turno 1 512= %x \n",vettoriSupporto[2053]);
					//printf("vettoriSupporto[18] prima delle somme= %x \n",vettoriSupporto[18]);
				} */
	overflowPari = 0;
	overflowDispari=0;
	if(t % 8 == 0 || t %8  ==4 ) { // ATTENZIONE, sostsituire risultato con risultatoTOT

		//offsetT1= t*2;
		//offsetT2= offsetT1 +2;
		for(k=0; k< 24; k++) {

			if(k <  4) {
				//risultato[t*4 + k] = vettoriSupporto[(t %8)*n + k];
			}
			if( k>= 4 && k < 20) {

				if(vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) +1] + vettoriSupporto[((t+2) %8)*n + k -4 + 4*(t-sottraggo1) +1] + overflowPari > 255)  {
					//overflowPari = 1;
					overflowDispari=1;
				} else {
					//overflowPari = 0;
					overflowDispari=0;
				}
				vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) +1]= vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) +1] + vettoriSupporto[((t+2) %8)*n + k -4 + 4*(t-sottraggo1) +1] +  overflowPari ;
				if(overflowDispari == 1) {
					overflowPari = 1;
				} else {
					overflowPari = 0;
				}

				/*vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) +1]= vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) +1] + vettoriSupporto[((t+2) %8)*n + k -4 + 4*(t-sottraggo1) +1] +  overflowPari ;
				if(vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) +1] + vettoriSupporto[((t+2) %8)*n + k -4 + 4*(t-sottraggo1) +1] + overflowPari > 255) {
					overflowPari = 1;
				} else {
					overflowPari = 0;
				} */

			}
			if( k >= 20) {

				if( vettoriSupporto[((t+2) %8)*n + k -4 + 4*(t-sottraggo1) +1] + overflowPari > 255) {
					//overflowPari = 1;
					overflowDispari=1;
				} else {
					//overflowPari = 0;
					overflowDispari=0;
				}
				vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) +1]=  vettoriSupporto[((t+2) %8)*n + k -4 + 4*(t-sottraggo1) +1] +  overflowPari ;
				if(overflowDispari == 1) {
					overflowPari = 1;
				} else {
					overflowPari = 0;
				}

				/*vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) +1]=  vettoriSupporto[((t+2) %8)*n + k -4 + 4*(t-sottraggo1) +1] +  overflowPari ;
				if( vettoriSupporto[((t+2) %8)*n + k -4 + 4*(t-sottraggo1) +1] + overflowPari > 255) {
					overflowPari = 1;
				} else {
					overflowPari = 0;
				} */

			}

		}
	} // fine turno somma 2
	__syncthreads();
	/*if( t== 0) {

			printf("vettoriSupporto[0] turno 2= %x \n",vettoriSupporto[0]);
			printf("vettoriSupporto[1] turno 2= %x \n",vettoriSupporto[1]);
			printf("vettoriSupporto[2] turno 2= %x \n",vettoriSupporto[2]);
			printf("vettoriSupporto[3] turno 2= %x \n",vettoriSupporto[3]);
			printf("vettoriSupporto[4] turno 2= %x \n",vettoriSupporto[4]);
			printf("vettoriSupporto[4] turno 2= %x \n",vettoriSupporto[5]);
			printf("vettoriSupporto[17] turno 2= %x \n",vettoriSupporto[17]);
			printf("vettoriSupporto[18] turno 2= %x \n",vettoriSupporto[18]);
		} */
	overflowPari = 0;
	overflowDispari=0;
	if(t % 8 == 0) {

		//offsetT1= t*2;
		//offsetT2= offsetT1 +2;
		for(k=0; k< 32; k++) {

			if(k <  8) {
				//vettoriSupporto[(t %4)*n + k] = vettoriSupporto[(t %4)*n + k];
				/*if(t== 544){
									printf("vettoreSupporto[%d] = %x \n",t*4 + k,vettoriSupporto[(t %4)*n + k]);
								} */
				risultato[t*4 + k] = vettoriSupporto[(t %8)*n + k];
				if(risultato[t*4 + k] != vettoriSupporto[(t %8)*n + k]) {
					printf("errore!");
				}
				/*if(t== 512){
					printf("risultato[%d] = %x \n",t*4 + k,risultato[t*4 + k]);
				} */
			}
			if( k>= 8 && k < 24) {

				/*if(t== 0 && k== 8) {

					printf("vettoriSupporto offeset 0 = %d \n",vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) +1]);
					printf("vettoriSupporto offset 4= %d \n",vettoriSupporto[((t+4) %8)*n + k -4 + 4*(t-sottraggo1) +1]);
				}
				if(t== 0 && k== 17) {

					printf("vettoriSupporto offeset 0 17= %d \n",vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) +1]);
					printf("vettoriSupporto offset 4 17= %d \n",vettoriSupporto[((t+4) %8)*n + k -4 + 4*(t-sottraggo1) +1]);
				} */

				if(vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) ] + vettoriSupporto[((t+4) %8)*n + k -4 + 4*(t-sottraggo1) ] + overflowPari > 255) {
								//overflowPari = 1;
								overflowDispari=1;
							} else {
								//overflowPari = 0;
								overflowDispari=0;
							}
				risultato[t*4 + k] = vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) ] + vettoriSupporto[((t+4) %8)*n + k -4 + 4*(t-sottraggo1) ] +  overflowPari ;
							if(overflowDispari == 1) {
								overflowPari = 1;
							} else {
								overflowPari = 0;
							}

				/*risultato[t*4 + k] = vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) ] + vettoriSupporto[((t+4) %8)*n + k -4 + 4*(t-sottraggo1) ] +  overflowPari ;
				if(vettoriSupporto[(t %8)*n + k + 4*(t-sottraggo1) ] + vettoriSupporto[((t+4) %8)*n + k -4 + 4*(t-sottraggo1) ] + overflowPari > 255) {
					overflowPari = 1;
				} else {
					overflowPari = 0;
				} */

			}
			if( k >= 24) {


				if(vettoriSupporto[((t+4) %8)*n + k -4 + 4*(t-sottraggo1) +1] + overflowPari > 255) {
					//overflowPari = 1;
					overflowDispari=1;
				} else {
					//overflowPari = 0;
					overflowDispari=0;
				}
				risultato[t*4 + k] = vettoriSupporto[((t+4) %8)*n + k -4 + 4*(t-sottraggo1) +1] +  overflowPari ;
				if(overflowDispari == 1) {
					overflowPari = 1;
				} else {
					overflowPari = 0;
				}

				/*risultato[t*4 + k] = vettoriSupporto[((t+4) %8)*n + k -4 + 4*(t-sottraggo1) +1] +  overflowPari ;
				if( vettoriSupporto[((t+4) %8)*n + k -4 + 4*(t-sottraggo1) +1] + overflowPari > 255) {
					overflowPari = 1;
				} else {
					overflowPari = 0;
				} */

			}

		}
	} // fine turno somma 3

	/*if(t==512) {
						printf("vettoriSupporto[0] turno 3 512= %x \n",vettoriSupporto[2048]);
						printf("vettoriSupporto[1] turno 3 512= %x \n",vettoriSupporto[2049]);
						printf("vettoriSupporto[2] turno 3 512= %x \n",vettoriSupporto[2050]);
						printf("vettoriSupporto[3] turno 3 512= %x \n",vettoriSupporto[2051]);
						printf("vettoriSupporto[4] turno 3 512= %x \n",vettoriSupporto[2052]);
						printf("vettoriSupporto[5] turno 3 512= %x \n",vettoriSupporto[2053]);
						printf("risultato[2048] turno 3 512= %x \n",risultato[2048]);
						printf("risultato[2049] turno 3 512= %x \n",risultato[2049]);
						printf("risultato[2050] turno 3 512= %x \n",risultato[2050]);
						//printf("vettoriSupporto[18] prima delle somme= %x \n",vettoriSupporto[18]);
	} */
	/*if( t== 0) {

				printf("vettoriSupporto[0] turno 3= %x \n",vettoriSupporto[0]);
				printf("vettoriSupporto[1] turno 3= %x \n",vettoriSupporto[1]);
				printf("vettoriSupporto[2] turno 3= %x \n",vettoriSupporto[2]);
				printf("vettoriSupporto[3] turno 3= %x \n",vettoriSupporto[3]);
				printf("vettoriSupporto[4] turno 3= %x \n",vettoriSupporto[4]);
				printf("vettoriSupporto[4] turno 3= %x \n",vettoriSupporto[5]);
				printf("vettoriSupporto[8] turno 3= %x \n",vettoriSupporto[8]);
				printf("vettoriSupporto[17] turno 3= %x \n",vettoriSupporto[17]);
				printf("vettoriSupporto[21] turno 3= %x \n",vettoriSupporto[21]);
				printf("risultato[0] turno 3= %x \n",risultato[0]);
				printf("risultato[1] turno 3= %x \n",risultato[1]);
				printf("risultato[2] turno 3= %x \n",risultato[2]);
				printf("risultato[3] turno 3= %x \n",risultato[3]);
				printf("risultato[4] turno 3= %x \n",risultato[4]);
				printf("risultato[8] turno 3= %x \n",risultato[8]);
				printf("risultato[17] turno 3= %x \n",risultato[17]);
				printf("risultato[21] turno 3= %x \n",risultato[21]);

			} */


}
__global__ void provacuda(void){


	int tid= blockIdx.x*blockDim.x+threadIdx.x;

	if(tid == 0 ) {

		printf("ciao2 \n");

	}
	printf("ciao \n");
}
__global__ void powerBigNumbersGPULevel3(unsigned char *risultato,unsigned char *vettoriSupporto, unsigned int esponente,unsigned int base, int *debug) {

	int t= blockIdx.x*blockDim.x+threadIdx.x;
	int supporto;
	int n =131070;
	int i;
	int j;
	int k;
	unsigned int riporto;
	unsigned int riporto16Pari;
	unsigned int riporto16Dispari;
	unsigned int overflow;
	unsigned int overflowDispari;
	unsigned int overflowPari;
	unsigned char supportoParteBassa;
	unsigned char riportoParteBassa;


	for(i=0;i< 16; i++) {

		base= risultato[t*64 + 32 +1 +i*2];
		base = base << 8;
		base= base + risultato[t*64 + 32 + i*2];

		if(i==0 && t==0){
			printf("base = %x \n",base);
		}

	 if(i==0){


		riporto=0;
		riporto16Pari=0;
		riporto16Dispari=0;
		overflow=0;
		overflowDispari=0;
		overflowPari=0;
	 for(j=0;j<32; j++) {
		 supporto = ((unsigned int) risultato[j + t*64]) * base;

		 if(t==0){
			 printf("supporto[%d] = %x \n",j,supporto);
		 }
		 supportoParteBassa= supporto;
		 riportoParteBassa = riporto;
		 if(j % 2 == 0) {

			 if( ((unsigned int) supportoParteBassa +  (unsigned int) riportoParteBassa) > 255 ) {
				 overflowPari=overflowPari +1;
			 }
			 //risultatoInt[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
			 //risultato[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
			 vettoriSupporto[n+ j + t*64 ] = (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
			 //if( ((unsigned int) risultato[j] +  (unsigned int) riporto16Pari) > 255 ) {
			 if( ((unsigned int) vettoriSupporto[n+ j + t*64  ] +  (unsigned int) riporto16Pari) > 255 ) {
				 overflowPari=overflowPari +1;
			 }
			 vettoriSupporto[n+ j + t*64  ] = (unsigned char) ( (unsigned int) vettoriSupporto[n+ j + t*64  ] + (unsigned int) riporto16Pari );

			 if( ((unsigned int) vettoriSupporto[j + t*64 ] +  (unsigned int) overflowDispari) > 255 ) {
				 overflowPari=overflowPari +1;
			 }
			 vettoriSupporto[n+ j + t*64  ] = (unsigned char) ( (unsigned int) vettoriSupporto[n+ j + t*64  ] + (unsigned int) overflowDispari );
			 overflowDispari=0;
		 } else {

			 if( ((unsigned int) supportoParteBassa +  (unsigned int) riportoParteBassa) > 255 ) {
				 overflowDispari=overflowDispari +1;
			 }
			 vettoriSupporto[n+ j + t*64 ] = (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );

			 if( ((unsigned int) vettoriSupporto[n+ j + t*64  ] +  (unsigned int) riporto16Dispari) > 255 ) {
				 overflowDispari=overflowDispari +1;
			 }
			 vettoriSupporto[n+ j + t*64 ] = (unsigned char) ( (unsigned int) vettoriSupporto[n+ j + t*64 ] + (unsigned int) riporto16Dispari );
			 if( ((unsigned int) vettoriSupporto[j + t*64  ] +  (unsigned int) overflowPari) > 255 ) {
				 overflowDispari=overflowDispari +1;
			 }
			 vettoriSupporto[ n+j + t*64  ] = (unsigned char) ( (unsigned int) vettoriSupporto[ n+ j + t*64  ] + (unsigned int) overflowPari );
			 overflowPari=0;

		 }
		 supporto= supporto >> 8;
		 riporto = supporto;
		 if(j % 2 == 0) {
			 riporto16Pari = riporto >> 8;
		 } else {
			 riporto16Dispari = riporto >> 8;

		 }

	 }

	 overflowPari=0;
	 riporto16Dispari=0;
	 	if( ((unsigned char) supporto  + (unsigned int) riporto16Pari) > 255 ) {
	 		overflowPari=1;
	 	}
	 	vettoriSupporto[n+ 32 + t*64] =  (unsigned char) supporto  + (unsigned int) riporto16Pari ;
	 	if( (unsigned int) vettoriSupporto[n+ 32 + t*64]  + (unsigned int) overflowDispari > 255 ) {
	 		overflowPari = overflowPari+ 1;
	 	}
	 	vettoriSupporto[n+ 32 + t*64] = (unsigned char) ( (unsigned int) vettoriSupporto[n+ 32 + t*64]  + (unsigned int) overflowDispari );
	 	supporto= supporto >> 8;
	 	vettoriSupporto[n+ 32 + t*64 +1] = (unsigned char) supporto  + (unsigned int) overflowPari ;
       if(t==0){
	 	for(k=0;k<34;k++){
	 		printf("vettoriSupporto[%d]= %x \n",k,vettoriSupporto[n+ k]);
	 	}
       }
	} else {

		riporto=0;
		riporto16Pari=0;
		riporto16Dispari=0;
		overflow=0;
		overflowDispari=0;
		overflowPari=0;
		for(j=0;j<32; j++) {
			supporto = ((unsigned int) risultato[j]) * base;
			supportoParteBassa= supporto;
			riportoParteBassa = riporto;

			if(j % 2 == 0) {

				if( ((unsigned int) supportoParteBassa +  (unsigned int) riportoParteBassa) > 255 ) {
					overflowPari=overflowPari +1;
				}
				//risultatoInt[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
				//risultato[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
				vettoriSupporto[ j + t*64 ] = (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
				//if( ((unsigned int) risultato[j] +  (unsigned int) riporto16Pari) > 255 ) {
				if( ((unsigned int) vettoriSupporto[j + t*64  ] +  (unsigned int) riporto16Pari) > 255 ) {
					overflowPari=overflowPari +1;
				}
				vettoriSupporto[j + t*64  ] = (unsigned char) ( (unsigned int) vettoriSupporto[j + t*64  ] + (unsigned int) riporto16Pari );

				if( ((unsigned int) vettoriSupporto[j + t*64 ] +  (unsigned int) overflowDispari) > 255 ) {
					overflowPari=overflowPari +1;
				}
				vettoriSupporto[j + t*64  ] = (unsigned char) ( (unsigned int) vettoriSupporto[j + t*64  ] + (unsigned int) overflowDispari );
				overflowDispari=0;
			} else {

				if( ((unsigned int) supportoParteBassa +  (unsigned int) riportoParteBassa) > 255 ) {
					overflowDispari=overflowDispari +1;
				}
				vettoriSupporto[j + t*64 ] = (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );

				if( ((unsigned int) vettoriSupporto[j + t*64  ] +  (unsigned int) riporto16Dispari) > 255 ) {
					overflowDispari=overflowDispari +1;
				}
				vettoriSupporto[j + t*64 ] = (unsigned char) ( (unsigned int) vettoriSupporto[j + t*64 ] + (unsigned int) riporto16Dispari );
				if( ((unsigned int) vettoriSupporto[j + t*64  ] +  (unsigned int) overflowPari) > 255 ) {
					overflowDispari=overflowDispari +1;
				}
				vettoriSupporto[j + t*64  ] = (unsigned char) ( (unsigned int) vettoriSupporto[j + t*64  ] + (unsigned int) overflowPari );
				overflowPari=0;

			}
			supporto= supporto >> 8;
			riporto = supporto;
			if(j % 2 == 0) {
				riporto16Pari = riporto >> 8;
			} else {
				riporto16Dispari = riporto >> 8;

			}

		}

		overflowPari=0;
		riporto16Dispari=0;
		if( ((unsigned char) supporto  + (unsigned int) riporto16Pari) > 255 ) {
			overflowPari=1;
		}
		vettoriSupporto[32 + t*64] =  (unsigned char) supporto  + (unsigned int) riporto16Pari ;
		if( (unsigned int) vettoriSupporto[32 + t*64]  + (unsigned int) overflowDispari > 255 ) {
			overflowPari = overflowPari+ 1;
		}
		vettoriSupporto[32 + t*64] = (unsigned char) ( (unsigned int) vettoriSupporto[32 + t*64]  + (unsigned int) overflowDispari );
		supporto= supporto >> 8;
		vettoriSupporto[32 + t*64 +1] = (unsigned char) supporto  + (unsigned int) overflowPari ;

		// qui devo sommare
		overflowPari=0;
		riporto16Dispari=0;
		for(j=0;j< 32+2*i +2; j++) {

			if(j < 2*i){

			}
			if(j>=2*i && j< 32+2*i ){

				if((vettoriSupporto[n+ j + t*64 ] + vettoriSupporto[ j + t*64 -2*i ] + overflowPari) > 255) {
					overflowDispari=1;
				} else {
					overflowDispari=0;
				}
				vettoriSupporto[n+ j + t*64 ]= vettoriSupporto[n+ j + t*64 ] + vettoriSupporto[ j + t*64 -2*i ] + overflowPari;
				if(overflowDispari == 1) {
				 overflowPari = 1;
				} else {
				 overflowPari = 0;
				}



			}
			if(j> 32+2*i  ){

				if(( vettoriSupporto[ j + t*64 -2*i ] + overflowPari) > 255) {
					overflowDispari=1;
				} else {
					overflowDispari=0;
				}
				vettoriSupporto[n+ j + t*64 ]= vettoriSupporto[ j + t*64 -2*i ] + overflowPari;
				if(overflowDispari == 1) {
					overflowPari = 1;
				} else {
					overflowPari = 0;
				}

			}

		}

	}// chiusura i==0


	}

	for(i=0;i<64;i++) {
		risultato[t*64 + i]= vettoriSupporto[n + t*64 +i];

	}
	//base= risultato(t + 32)



}
//int powerBigNumbersGPU(unsigned char *risultato,unsigned int esponente,unsigned int base);
int main(void)
{
	hipEvent_t event;
	hipDeviceReset();
   // unsigned int esponente; // numero massimo su 16 bit, deve essere almeno 2
    // unsigned int base;  // numero massimo su 16 bit
     int n =131070;
     unsigned char *dev_vect;
     unsigned char *vect;
     unsigned char *dev_variabili;
     unsigned char *variabili;
     int i;
     int *dev_debug;
     int *debug;

     hipEventCreateWithFlags(&event,hipEventDisableTiming);
     debug = (int*) malloc(sizeof(int));
         hipMalloc((void**)&dev_debug,sizeof(int));
         hipMemcpy(dev_debug,debug,sizeof(int),hipMemcpyHostToDevice);

     variabili = (unsigned char*) malloc(n*9*sizeof(unsigned char));  // in realta variabili è una matrice srotolata

     hipMalloc((void**)&dev_variabili,9*n*sizeof(unsigned char));
     hipMemcpy(dev_variabili,variabili,9*n*sizeof(unsigned char),hipMemcpyHostToDevice);

     vect = (unsigned char*) malloc(n*sizeof(unsigned char));
     hipMalloc((void**)&dev_vect,n*sizeof(unsigned char));
     hipMemcpy(dev_vect,vect,n*sizeof(unsigned char),hipMemcpyHostToDevice);

	// il numero massimo di moltiplicazioni è 65535, ogni thread gestisce 2 moltiplicazioni
	// quindi il numero di thread necessari è 32
     provacuda<<<1,10>>>();
	//memoriaGlobale<<<64,1024>>>(dev_vect, n,numeroBlocchi);
	powerBigNumbersGPUShared<<<64,512>>>(dev_vect,dev_variabili, 4096,7,dev_debug);
	cudaCheckErrors("hipMemcpy 1 fail");
	/*cudaMemcpy(vect, dev_vect, n*sizeof(unsigned char), cudaMemcpyDeviceToHost);
	for(i=0;i<4096;i++) {
		if(i %8 == 0) {
			if(vect[i] != 97) {
				printf("errMain doveva essere 97 step1, risultato[%d]= %d \n ",i,vect[i]);
			}
		}
		if(i %8 == 1) {
			if(vect[i] != 9) {
				printf("errMain doveva essere 9 step1,risultato[%d]= %d \n ",i,vect[i]);
			}

		} else {
			if(vect[i] != 0 && i %8 != 0) {
				printf("errMain doveva essere 0 step1, risultato[%d]= %d \n ",i,vect[i]);
			}
		}
	}
	cudaMemcpy(dev_vect,vect,n*sizeof(unsigned char),cudaMemcpyHostToDevice); */
	powerBigNumbersGPUStep2<<<64,512>>>(dev_vect,dev_variabili, 4096,7,dev_debug);
	cudaCheckErrors("hipMemcpy 1 fail");
	/*cudaMemcpy(vect, dev_vect, n*sizeof(unsigned char), cudaMemcpyDeviceToHost);
	for(i=0;i<4096;i++) {
		switch (i%16) {
				case 0 :
					if(vect[i] != 193) {
						printf("errMain, doveva essere 193 step2, risultato[%d]= %d \n",i,vect[i]);

					}
					break;
				case 1:
					if(vect[i] != 246) {
						printf("errMain, doveva essere 246 step2, risultato[%d]= %d \n",i,vect[i]);

					}
					break;
				case 2:

					if(vect[i] != 87) {
						printf("errMain, doveva essere 87 step2, risultato[%d]= %d \n",i,vect[i]);

					}
					break;
				default:
					if(vect[i] != 0) {
						printf("errMain, doveva essere 0 step2, risultato[%d]= %d \n",i,vect[i]);
					}
			    break;

				}
	}
	cudaMemcpy(dev_vect,vect,n*sizeof(unsigned char),cudaMemcpyHostToDevice); */
	powerBigNumbersGPUStep3<<<64,512>>>(dev_vect,dev_variabili, 4096,7,dev_debug);
	cudaCheckErrors("hipMemcpy 1 fail");
	hipEventRecord(event);
	while(hipEventQuery(event) != hipSuccess)
	{
		//dostuff
	}
	//cudaDeviceSynchronize();
	hipMemcpy(vect, dev_vect, n*sizeof(unsigned char), hipMemcpyDeviceToHost);
	for(i=0;i<4096;i++) {
			switch (i%32) {
					case 0 :
						if(vect[i] != 129) {
							printf("errMain, doveva essere 129 step3, risultato[%d]= %d \n",i,vect[i]);

						}
						break;
					case 1:
						if(vect[i] != 125) {
							printf("errMain, doveva essere 125 step3, risultato[%d]= %d \n",i,vect[i]);

						}
						break;
					case 2:

						if(vect[i] != 5) {
							printf("errMain, doveva essere 5 step3, risultato[%d]= %d \n",i,vect[i]);

						}
						break;
					case 3:

						if(vect[i] != 165) {
							printf("errMain, doveva essere 165 step3, risultato[%d]= %d \n",i,vect[i]);

						}
						break;
					case 4:

						if(vect[i] != 57) {
							printf("errMain, doveva essere 57 step3, risultato[%d]= %d \n",i,vect[i]);

						}
						break;
					case 5:

						if(vect[i] != 30) {
							printf("errMain, doveva essere 30 step3, risultato[%d]= %d \n",i,vect[i]);

						}
						break;
					default:
						if(vect[i] != 0) {
							printf("errMain, doveva essere 0 step3, risultato[%d]= %d \n",i,vect[i]);
						}
				    break;

					}
		}
	printf(" risultato[0]= %x \n",vect[0]);
	printf(" risultato[1]= %x \n",vect[1]);
	printf(" risultato[2]= %x \n",vect[2]);
	printf(" risultato[3]= %x \n",vect[3]);
	printf(" risultato[4]= %x \n",vect[4]);
	printf(" risultato[5]= %x \n",vect[5]);
	printf(" risultato[5]= %x \n",vect[6]);
	hipMemcpy(dev_vect,vect,n*sizeof(unsigned char),hipMemcpyHostToDevice);
	powerBigNumbersGPULevel3<<<1,64>>>(dev_vect,dev_variabili, 4096,7,dev_debug);
	cudaCheckErrors("hipMemcpy 1 fail");
	hipMemcpy(vect, dev_vect, n*sizeof(unsigned char), hipMemcpyDeviceToHost);
	printf(" risultato[0] level3= %x \n",vect[0]);
	printf(" risultato[1]  level3= %x \n",vect[1]);
	printf(" risultato[2]  level3= %x \n",vect[2]);
	printf(" risultato[3]  level3= %x \n",vect[3]);
	printf(" risultato[4]  level3= %x \n",vect[4]);
	printf(" risultato[5]  level3= %x \n",vect[5]);
	printf(" risultato[6] level3= %x \n",vect[6]);
	printf(" risultato[7]  level3= %x \n",vect[7]);
	printf(" risultato[8]  level3= %x \n",vect[8]);
	printf(" risultato[9] level3= %x \n",vect[9]);
	printf(" risultato[10] level3= %x \n",vect[10]);
	printf(" risultato[11] level3= %x \n",vect[11]);
	printf(" risultato[12] level3= %x \n",vect[12]);
	for(i=0;i<4096;i++) {
				switch (i%64) {
						case 0 :
							if(vect[i] != 1) {
								printf("errMain, doveva essere 1 step3, risultato[%d]= %d \n",i,vect[i]);

							}
							break;
						case 1:
							if(vect[i] != 59) {
								printf("errMain, doveva essere 59 step3, risultato[%d]= %d \n",i,vect[i]);

							}
							break;
						case 2:

							if(vect[i] != 145) {
								printf("errMain, doveva essere 145 step3, risultato[%d]= %d \n",i,vect[i]);

							}
							break;
						case 3:

							if(vect[i] != 110) {
								printf("errMain, doveva essere 110 step3, risultato[%d]= %d \n",i,vect[i]);

							}
							break;
						case 4:

							if(vect[i] != 88) {
								printf("errMain, doveva essere 88 step3, risultato[%d]= %d \n",i,vect[i]);

							}
							break;
						case 5:

							if(vect[i] != 51) {
								printf("errMain, doveva essere 51 step3, risultato[%d]= %d \n",i,vect[i]);

							}
							break;
						case 6:

							if(vect[i] != 60) {
								printf("errMain, doveva essere 60 step3, risultato[%d]= %d \n",i,vect[i]);

							}
							break;
						case 7:

							if(vect[i] != 48) {
								printf("errMain, doveva essere 48 step3, risultato[%d]= %d \n",i,vect[i]);

							}
							break;
						case 8:

							if(vect[i] != 168) {
								printf("errMain, doveva essere 168 step3, risultato[%d]= %d \n",i,vect[i]);

							}
							break;
						case 9:

							if(vect[i] != 143) {
								printf("errMain, doveva essere 143 step3, risultato[%d]= %d \n",i,vect[i]);

							}
							break;
						case 10:

							if(vect[i] != 145) {
								printf("errMain, doveva essere 145 step3, risultato[%d]= %d \n",i,vect[i]);

							}
							break;
						case 11:

							if(vect[i] != 3) {
								printf("errMain, doveva essere 145 step3, risultato[%d]= %d \n",i,vect[i]);

							}
							break;
						default:
							if(vect[i] != 0) {
								printf("errMain, doveva essere 0 step3, risultato[%d]= %d \n",i,vect[i]);
							}
					    break;

						}
			}
	/*for(i=0; i< 4096; i++) {


		if(i%16 == 0) {
			if(vect[i] != 193) {
				printf("errMain doveva essere 193, risultato[%d]= %d \n ",i,vect[i]);
			}

		}
		if(i%16 == 1) {
			if(vect[i] != 246) {
				printf("errMain doveva essere 246, risultato[%d]= %d \n ",i,vect[i]);
			}

	    }
		if(i%16 == 2) {
			if(vect[i] != 87) {
				printf("errMain doveva essere 87, risultato[%d]= %d \n ",i,vect[i]);
			}

	    }
		if(i%16 != 0 && i%16 != 1 && i%16 != 2  ) {
			if(vect[i] != 0) {
				printf("errMain doveva essere 0, risultato[%d]= %d \n ",i,vect[i]);
			}

		}

		   // test i=1
		if(i %8 == 0) {
			if(vect[i] != 97) {
				printf("errMain doveva essere 97, risultato[%d]= %d \n ",i,vect[i]);
			}
		}
		if(i %8 == 1) {
			if(vect[i] != 9) {
				printf("errMain doveva essere 9,risultato[%d]= %d \n ",i,vect[i]);
			}

		} else {
			if(vect[i] != 0 && i %8 != 0) {
				printf("errMain doveva essere 0, risultato[%d]= %d \n ",i,vect[i]);
			}
		}


		switch (i%16) {
		case 0 :
			if(vect[i] != 193) {
				printf("errMain, doveva essere 193, risultato[%d]= %d \n",i,vect[i]);

			}
			break;
		case 1:
			if(vect[i] != 246) {
				printf("errMain, doveva essere 246, risultato[%d]= %d \n",i,vect[i]);

			}
			break;
		case 2:

			if(vect[i] != 87) {
				printf("errMain, doveva essere 87, risultato[%d]= %d \n",i,vect[i]);

			}
			break;
		default:
			if(vect[i] != 0) {
				printf("errMain, doveva essere 0, risultato[%d]= %d \n",i,vect[i]);
			}
	    break;

		}



	}
     */
	hipDeviceReset();
		  return 0;
	// 7^15 = 45160B7A437
	/*
  const int n = 1048576;
  int numeroBlocchi = 10;
  int *giapassati ;
  int *giapassatihost;
  int i;
  //double  a[n], r[n], d[n];


  double *vect;
  double *dev_vect;

  double *vect2;
  double *dev_vect2;

  vect = (double*) malloc(n*sizeof(double));
  cudaMalloc((void**)&dev_vect,n*sizeof(double));
  cudaMemcpy(dev_vect,vect,n*sizeof(double),cudaMemcpyHostToDevice);
 // cudaMalloc(&d_d, n * sizeof(double));
  // run version with static shared memory
  //cudaMemcpy(d_d, a, n*sizeof(double), cudaMemcpyHostToDevice);


  memoriaGlobale<<<64,1024>>>(dev_vect, n,numeroBlocchi);
  cudaMemcpy(vect, dev_vect, n*sizeof(double), cudaMemcpyDeviceToHost);
  for (int i = 0; i < n; i++) {
	  if(vect[i] != i) {

		  printf("errore 1 ! \n" );
		  printf("d[%i] = %f \n",i,vect[i]);
	  }
	  if(i== 0) {
	      printf("d[%i] = %f \n",i,vect[i]);
	  	  }
	  if(i== 1048575) {
    printf("d[%i] = %f \n",i,vect[i]);
	  }
  }



  cudaMalloc((void**)&giapassati,384*sizeof(int));
  vect2 = (double*) malloc(n*sizeof(double));
  giapassatihost = (int*) malloc(384*sizeof(int));
  for(i=0; i< 384 ; i++) {

    	  		giapassatihost[i] = 0;

    	  	}
  cudaMemcpy(giapassati,giapassatihost,384*sizeof(int),cudaMemcpyHostToDevice);
   cudaMalloc((void**)&dev_vect2,n*sizeof(double));
   cudaMemcpy(dev_vect2,vect2,n*sizeof(double),cudaMemcpyHostToDevice);
  // cudaMalloc(&d_d, n * sizeof(double));
   // run version with static shared memory
   //cudaMemcpy(d_d, a, n*sizeof(double), cudaMemcpyHostToDevice);

   for(i=0; i< 384 ; i++) {

  	  		//giapassati[i] = 0;

  	  	}

   memoriaShared<<<1,385>>>(dev_vect2, giapassati);
   cudaMemcpy(vect2, dev_vect2, n*sizeof(double), cudaMemcpyDeviceToHost);
   for (int i = 0; i < n; i++) {
	   if(vect2[i] != i) {

		   printf("errore 2 ! \n" );
		   printf("d[%i] = %f \n",i,vect2[i]);
	   }
	  // printf("d[%i] = %f \n",i,vect2[i]);
 	  if(i== 1048575) {
     printf("d[%i] = %f \n",i,vect2[i]);
 	  }
   }
  // run dynamic shared memory version
 // cudaMemcpy(d_d, a, n*sizeof(int), cudaMemcpyHostToDevice);
 // dynamicReverse<<<1,n,n*sizeof(int)>>>(d_d, n);
 // cudaMemcpy(d, d_d, n * sizeof(int), cudaMemcpyDeviceToHost);
 // for (int i = 0; i < n; i++)
  //  if (d[i] != r[i]) printf("Error: d[%d]!=r[%d] (%d, %d)n", i, i, d[i], r[i]);
  */

}






