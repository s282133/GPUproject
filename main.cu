#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
//#include <math.h>
/*
__global__ void memoriaGlobale(double *d, int n, int numeroBlocchi)
{
 // __shared__ int s[n];
	int i;

  int t = (blockIdx.x*blockDim.x)+threadIdx.x;
  //printf("t : %d \n",t);
  //int tr = n-t-1;
  //d[t] = t ;
   // printf("t : %d \n", t);
   d[t*16] = t*16 ;
   d[t*16+1] = t*16+1 ;
   d[t*16+2] = t*16+2 ;
   d[t*16+3] = t*16+3 ;
   d[t*16+4] = t*16+4 ;
   d[t*16+5] = t*16+5 ;
   d[t*16+6] = t*16+6 ;
   d[t*16+7] = t*16+7 ;
   d[t*16+8] = t*16+8 ;
   d[t*16+9] = t*16+9 ;
   d[t*16+10] = t*16+10 ;
   d[t*16+11] = t*16+11 ;
   d[t*16+12] = t*16+12 ;
   d[t*16+13] = t*16+13 ;
   d[t*16+14] = t*16+14 ;
   d[t*16+15] = t*16+15 ;
   //printf("d : %f \n", d[t*16+15]);
   printf("error \n");
   for(i=0; i< 16; i++ ) {

	   if(d[t*16+i] != t*16+i) {
		   printf("error \n");
	   }
   }
  // __syncthreads();
  // d[t] = s[tr];
}

__global__ void memoriaShared(double *d, int *giapassati)
{
  //extern __shared__ int s[];
	const int dimMEM = 6144; // perche 6144 è la dimewnsione massima
	const int blocchiDa16 = 100; // 6144/16; // da 16 perche ogni thread lavora su 16 dati
	__shared__ double s[dimMEM];
	//int permessi[blocchiDa16];
	int tArrotondato;
	int i;
	int var=1;
	int contagiapassati=0;
	int controllo=0;


  int t = (blockIdx.x*blockDim.x)+threadIdx.x;
 // if(t < blocchiDa16) {

	//  printf("t generati minori di 100 : %d \n",t);
  //}
  //printf("\n");
  //printf("\n");


   if(t == 97 ) {

  	   printf("ciao sono 97 prima \n ");
     }


   tArrotondato= t % (blocchiDa16);

   if(t == 97 ) {

     	   printf("ciao sono 97 dopo \n tArrotondato = %d \n", tArrotondato);
        }
   //__syncthreads();
  while(t >= blocchiDa16*var) {
	  if(t == 97 ) {

	  	   printf("ciao sono 97 in while \n ");
	     }
  }
   if(t == 97 ) {

	   printf("ciao sono 97 dopo dopo \n ");
   }

  giapassati[tArrotondato] = giapassati[tArrotondato] +1;
  printf("t che è riuscito a passare : %d, contagiapassati : %d \n",tArrotondato, giapassati[tArrotondato]);
   //permessi[tArrotondato] =1;


  s[tArrotondato*16] = t*16;
  s[tArrotondato*16 +1] = t*16 + 1;
  s[tArrotondato*16 +2] = t*16 + 2;
  s[tArrotondato*16 +3] = t*16 + 3;
  s[tArrotondato*16 +4] = t*16 + 4;
  s[tArrotondato*16 +5] = t*16 + 5;
  s[tArrotondato*16 +6] = t*16 + 6;
  s[tArrotondato*16 +7] = t*16 + 7;
  s[tArrotondato*16 +8] = t*16 + 8;
  s[tArrotondato*16 +9] = t*16 + 9;
  s[tArrotondato*16 +10] = t*16 + 10;
  s[tArrotondato*16 +11] = t*16 + 11;
  s[tArrotondato*16 +12] = t*16 + 12;
  s[tArrotondato*16 +13] = t*16 + 13;
  s[tArrotondato*16 +14] = t*16 + 14;
  s[tArrotondato*16 +15] = t*16 + 15;
   // __syncthreads();


  d[t*16] = s[tArrotondato*16];
  d[t*16+1] = s[tArrotondato*16+1];
  d[t*16+2] = s[tArrotondato*16+2];
  d[t*16+3] = s[tArrotondato*16+3];
  d[t*16+4] = s[tArrotondato*16+4];
  d[t*16+5] = s[tArrotondato*16+5];
  d[t*16+6] = s[tArrotondato*16+6];
  d[t*16+7] = s[tArrotondato*16+7];
  d[t*16+8] = s[tArrotondato*16+8];
  d[t*16+9] = s[tArrotondato*16+9];
  d[t*16+10] = s[tArrotondato*16+10];
  d[t*16+11] = s[tArrotondato*16+11];
  d[t*16+12] = s[tArrotondato*16+12];
  d[t*16+13] = s[tArrotondato*16+13];
  d[t*16+14] = s[tArrotondato*16+14];
  d[t*16+15] = s[tArrotondato*16+15];


 // permessi[tArrotondato] =0;
   if(t  == blocchiDa16*var -1) {
       controllo=0;



       while(controllo == 0) {
    	   controllo=1;
    	   for(i=0;i< blocchiDa16; i++) {

    		   if(giapassati[i] ==  0) {
    			   controllo = 0;
    			 //  printf("giapassati[%d] è ancora a zero \n",i);
    		   }
    	   }

       }

	   //var= var+1;
	   printf("sono T  = %d , var = %d, dovrei incrementare var \n",t,var);
   }
}

*/

__global__ void powerBigNumbersGPUShared(unsigned char *risultato,int *vettoriSupporto, unsigned int esponente,unsigned int base) {

	//__shared__ unsigned char risultatoTot[4096];
	//__shared__ unsigned char vettoriSupporto[9][4096];
	int n =131070;
	int t= blockIdx.x*blockDim.x+threadIdx.x;
	int dato1;
    int dato2;
    int risultatoStep;
    int threadUtili;
    int potenze2;
    int contapotenze;
    unsigned char parteBassaBase;
    unsigned char parteAltaBase;
    int i;
    int j;
    int k;
    int passi;
    int sommo2;
    int sottraggo1;
    unsigned int supporto;
    unsigned int riporto;
    unsigned int riporto16Pari;
    unsigned int riporto16Dispari;
    unsigned int overflow;
    unsigned int overflowPari;
    unsigned int overflowDispari;
    unsigned char supportoParteBassa;
    unsigned char riportoParteBassa;
   // risultato[0]=0;
    //risultato[1]=0;

   // dato1= risultato[0];
   // dato2 = risultato[1];
   // moltiplicazione= dato1*dato2;

	/*if(t == 0) {

		printf("ciaojsdnk \n");
		printf("moltiplicazione = %d \n",moltiplicazione);
	} */

   // printf("variabili 00 = %d \n",variabili[0]);
   // printf("variabili 00 = %d \n",variabili[1]);
	threadUtili = esponente /2;
		if(esponente % 2 == 1) {
			threadUtili++;
		}
		//debug
		/*if(t== 0){

			printf("thread utili = %d \n",threadUtili);
		} */
		//fine debug

		// la funzionew log non va
		potenze2=2;
		contapotenze=1;
		i=0;
		while(i== 0) {

			if(esponente < potenze2) {
				i=1;
			} else {
				potenze2=potenze2*2;
				contapotenze++;
			}
		}


		//risultatoLog = log2(esponente);
		//risultatoLogInt = log2(esponente);
		/*if(risultatoLog > risultatoLogInt) {

			risultatoLogInt = risultatoLogInt +1;
		} */
		passi=contapotenze ;
		//debug
			if(t== 0){

				printf("pasi utili = %d \n",passi);
			}
			//fine debug
			parteBassaBase = base;
			base = base >> 8;
			parteAltaBase = base;
			//debug
			if(t== 0){

						printf("parte bassa base = %x \n",parteBassaBase);
						printf("parte alta base = %x \n",parteAltaBase);
		     }

			// inizializzazione vettore con tutte le basi

					if(t < threadUtili - 1) {
					risultato[t*4] = parteBassaBase;
					risultato[t*4 +1] = parteAltaBase;
					risultato[t*4 +2] = parteBassaBase;
					risultato[t*4 +3] = parteAltaBase;
					} else {

						risultato[t*4] = 0;
					    risultato[t*4 +1] = 0;
						risultato[t*4 +2] = 0;
					    risultato[t*4 +3] = 0;
					}

					if(t == threadUtili - 1) {

						if(esponente % 2 == 1) { // se l'esponente è dispari un thread ha solo mezzo dato
							risultato[t*4] = parteBassaBase;
							risultato[t*4 +1] = parteAltaBase;
							risultato[t*4 +2] = 0;
							risultato[t*4 +3] = 0;

						} else { // se l'esponente è pari è tutto ok

							risultato[t*4] = parteBassaBase;
							risultato[t*4 +1] = parteAltaBase;
							risultato[t*4 +2] = parteBassaBase;
							risultato[t*4 +3] = parteAltaBase;

						}
					}
			       __syncthreads();
			//fine debug

	   if(t < threadUtili ) {


		for(i=0; i < passi; i++) {

			if(i== 0){

			    	 dato1= risultato[t*4 +1];
			    	 dato1 = dato1 << 8;
			    	 dato1= dato1 + risultato[t*4];
			    	 dato2= risultato[t*4 +3];
			    	 dato2 = dato2 << 8;
			    	 dato2= dato2 + risultato[t*4 +2];
			    	 //dato1 = dato1*dato2; a quanto pare se non riuso dato1 non funziona, e non so il perche
			    	 if(t== 0){

			    		 printf("dato1 = %d \n", dato1);
			    		 printf("dato2 = %d \n", dato2);
			    	 }
			    	 if(dato2 == 0) {

			    		 risultatoStep= dato1;
			    		 risultato[t*4] = risultatoStep;
			    		 //risultatoTot[t*4] = risultatoStep;
			    		 risultatoStep = risultatoStep >>8;
			    		 risultato[t*4 +1 ] = risultatoStep;
			    		 //risultatoTot[t*4 +1] = risultatoStep;
			    		 risultatoStep = risultatoStep >>8;
			    		 risultato[t*4 +2 ] = risultatoStep;
			    		// risultatoTot[t*4 +2] = risultatoStep;
			    		 risultatoStep = risultatoStep >>8;
			    		 risultato[t*4 +3 ] = risultatoStep;
			    		 //risultatoTot[t*4 +3] = risultatoStep;

			    	 } else {

			            	// dato1 = ((unsigned int)dato1)+((unsigned int)dato2);
			             risultatoStep= dato1 *dato2;

			            	 //printf("dato2 = %d \n", dato1);
			    		 risultato[t*4] = risultatoStep;
			    		 //risultatoTot[t*4 ] = risultatoStep;
			    		 risultatoStep = risultatoStep >>8;
			    		 risultato[t*4 +1 ] = risultatoStep;
			    		 //risultatoTot[t*4 +1] = risultatoStep;
			    		 risultatoStep = risultatoStep >>8;
			    		 risultato[t*4 +2 ] = risultatoStep;
			    		 //risultatoTot[t*4 +2] = risultatoStep;
			    		 risultatoStep = risultatoStep >>8;
			    		 risultato[t*4 +3 ] = risultatoStep;
			    		 //risultatoTot[t*4 +3] = risultatoStep;

			    	 }



			     } // fine i=0
			if(i== 1) {
				__syncthreads();
				if(t == 0) {
					for(j=0; j< 30; j++) {
					printf("risultato[%d] = %d \n",j,risultato[j]);
					}
				}

				 if(t % 2 == 0) {
				    sommo2=0;
				    sottraggo1=0;

				 } else { // servono a far si che i thread dispari puntino al posto giusto
					 sommo2=2;
				     sottraggo1=1;
				 }
				    // devo reimpostare base
				    base = risultato[t*8 +4 +1 + sommo2] ;
				    base = base << 8;
				    base = base + risultato[t*8 +4 + sommo2] ;
				    riporto=0;
				    riporto16Pari=0;
				    riporto16Dispari=0;
				    overflow=0;
				    overflowDispari=0;
				    overflowPari=0;
				    		  if(t== 0) {

				    			  printf("base = %d \n",base);
				    		  }

				  for(j=8*(t-sottraggo1); j< 4 + 8*(t-sottraggo1);j++) {

				    supporto = ((unsigned int) risultato[j]) * base;
				    supportoParteBassa= supporto;
				    riportoParteBassa = riporto;

				    if(j % 2 == 0) {

				      if( ((unsigned int) supportoParteBassa +  (unsigned int) riportoParteBassa) > 255 ) {
				    	overflowPari=overflowPari +1;
				      }
				       //risultatoInt[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
				    		      			        //risultato[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
				      vettoriSupporto[(t %2)*n + j] = (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
				    		      			     //if( ((unsigned int) risultato[j] +  (unsigned int) riporto16Pari) > 255 ) {
				      if( ((unsigned int) vettoriSupporto[(t %2)*n + j] +  (unsigned int) riporto16Pari) > 255 ) {
				    	overflowPari=overflowPari +1;
				      }
				    		      			     // risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) riporto16Pari );
				      vettoriSupporto[(t %2)*n + j] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %2)*n + j] + (unsigned int) riporto16Pari );

				      if( ((unsigned int) vettoriSupporto[(t %2)*n + j] +  (unsigned int) overflowDispari) > 255 ) {
				        overflowPari=overflowPari +1;
				      }
				    		      			     // risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) overflowDispari );
				      vettoriSupporto[(t %2)*n + j] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %2)*n + j] + (unsigned int) overflowDispari );
				      overflowDispari=0;
				    		      			                 // risultato[j]= (char) ( (unsigned int) supporto + (unsigned int) riporto + (unsigned int) riporto16Pari);
				      } else {

				      if( ((unsigned int) supportoParteBassa +  (unsigned int) riportoParteBassa) > 255 ) {
				       overflowDispari=overflowDispari +1;
				      }
				    		      			     // risultato[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
				      vettoriSupporto[(t %2)*n + j] = (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );

				      if( ((unsigned int) vettoriSupporto[(t %2)*n + j] +  (unsigned int) riporto16Dispari) > 255 ) {
				       overflowDispari=overflowDispari +1;
				      }
				    		      			    // risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) riporto16Dispari );
				      vettoriSupporto[(t %2)*n + j] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %2)*n + j] + (unsigned int) riporto16Dispari );
				      if( ((unsigned int) vettoriSupporto[(t %2)*n + j] +  (unsigned int) overflowPari) > 255 ) {
				       overflowDispari=overflowDispari +1;
				      }
				    		      			     //risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) overflowPari );
				      vettoriSupporto[(t %2)*n + j] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %2)*n + j] + (unsigned int) overflowPari );
				      overflowPari=0;

				    		      			                  //risultato[j]= (char) ( (unsigned int) supporto + (unsigned int) riporto + (unsigned int) riporto16PDispari);
				      }
				      supporto= supporto >> 8;

				    		      			                 // supporto = supporto + overflow;
				    		      			                 // overflow=0;
				      riporto = supporto;
				      if(j % 2 == 0) {
				       riporto16Pari = riporto >> 8;
				      } else {
				       riporto16Dispari = riporto >> 8;

				      }
				    } // chiusura for
				    overflowPari=0;
				    riporto16Dispari=0;
				    		      		  // printf("riporto16Pari = %x \n",riporto16Pari);
				    		      		  //printf("supporto = %x \n",supporto);
				    		      		  //printf("overflowDispari = %x \n",overflowDispari);
				    if( ((unsigned char) supporto  + (unsigned int) riporto16Pari) > 255 ) {
				     overflowPari=1;
				    		      		               //   printf("overflow viene aumentato 1 \n");
				    }
				    		      		  // 8*(t-sottraggo1) serve a fare in modo che ad esempio il thread 2 punti a partire da 8 celle in piu, in modo da saltare thread 0 e 1
				    vettoriSupporto[(t %2)*n + 4 + 8*(t-sottraggo1)] =  (unsigned char) supporto  + (unsigned int) riporto16Pari ;

				    if( (unsigned int) vettoriSupporto[(t %2)*n + 4 + 8*(t-sottraggo1)]  + (unsigned int) overflowDispari > 255 ) {
				     overflowPari = overflowPari+ 1;
				    		      		               // printf("overflow viene aumentato 2 \n");
				    }
				    		      		  //risultato[2 + 2*i] = (unsigned char) ( (unsigned int) risultato[2 + 2*i] + (unsigned int) overflowDispari );
				    vettoriSupporto[(t %2)*n+ 4 + 8*(t-sottraggo1)] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %2)*n+ 4 + 8*(t-sottraggo1)]  + (unsigned int) overflowDispari );

				    supporto= supporto >> 8;
				    		      		  //risultato[2 + 2*i +1] = (unsigned char) supporto  + (unsigned int) overflowPari ;
				    vettoriSupporto[(t %2)*n+ 4 + 8*(t-sottraggo1) +1] = (unsigned char) supporto  + (unsigned int) overflowPari ;

				    		      		 // }

				    __syncthreads();

				    		      		  // i thread pari sommano i due risultati per ora in risultato, riciclo overflow pari e overflow dispari
				    overflowPari = 0;
				    overflowDispari=0;
				    if(t % 2 == 0) {  // ATTENZIONE, sostsituire risultato con risultatoTOT

				     risultato[t*8] = vettoriSupporto[t*8];
				     risultato[t*8 +1] = vettoriSupporto[t*8 +1];
				    if(vettoriSupporto[t*8 +2] + vettoriSupporto[n+ t*8] > 255) {
				     overflowPari = 1;
				    }
				    risultato[t*8 +2] = vettoriSupporto[t*8 +2] + vettoriSupporto[n+ t*8];
				    if(vettoriSupporto[t*8 +3] + vettoriSupporto[n+ t*8+1 ] + overflowPari > 255) {
				     overflowDispari = 1;
				    }
				    risultato[t*8 +3] = vettoriSupporto[t*8 +3] + vettoriSupporto[n + t*8 +1 ] +overflowPari;
				    if(vettoriSupporto[t*8 +4] + vettoriSupporto[n + t*8 +2 ] + overflowDispari  > 255) {
				     overflowPari = 1;
				    }
				    risultato[t*8 +4] = vettoriSupporto[t*8 +4] + vettoriSupporto[n+ t*8 +2]+ overflowDispari;
				    if(vettoriSupporto[t*8 +5] + vettoriSupporto[n + t*8+3 ] + overflowPari > 255) {
				     overflowDispari = 1;
				    }
				    risultato[t*8 +5] = vettoriSupporto[t*8 +5] + vettoriSupporto[n + t*8 +3 ] +overflowPari;
				    if(vettoriSupporto[t*8 +6] + vettoriSupporto[n + t*8 +4 ] + overflowDispari  > 255) {
				     overflowPari = 1;
				    }
				    risultato[t*8 +6] =  vettoriSupporto[n + t*8 +4]+ overflowDispari; // se ci sono problemi controllare qui
				    risultato[t*8 +7] =  vettoriSupporto[n + t*8 +5 ] +overflowPari;
				   }
				    __syncthreads();
				    if(t== 0) {

				     for(k=0; k < 7; k++ ) {

				      printf(" hghj risultato[%d] = %x \n",k,risultato[k] );
				      printf("ghv risultato[%d] = %x \n",k+1024,risultato[k+1024] );
				     }


				    }



			} // fine i=1, fino a qui va

			if(i==2) {

				__syncthreads();
				if(t == 0) {
				 for(j=0; j< 30; j++) {
				  printf("risultato[%d] = %d \n",j,risultato[j]);
				  }
				}

			    if(t % 4 == 0) {
				 sommo2=0;
				 sottraggo1=0;
                }
			    if(t % 4 == 1) {
			     sommo2=2;
			     sottraggo1=1;
			    }
			    if(t % 4 == 2) {
			     sommo2=4;
			     sottraggo1=2;
			    }
			    if(t % 4 == 3) {
			     sommo2=6;
			     sottraggo1=3;
			    }

								    // devo reimpostare base
				 base = risultato[t*8 +4 +1 + sommo2] ;
			     base = base << 8;
				 base = base + risultato[t*8 +4 + sommo2] ;
				 riporto=0;
				 riporto16Pari=0;
				 riporto16Dispari=0;
				 overflow=0;
				 overflowDispari=0;
				 overflowPari=0;
				if(t== 0) {

				  printf("base = %d \n",base);
				}

				for(j=16*(t-sottraggo1); j< 8 + 16*(t-sottraggo1);j++) {

					supporto = ((unsigned int) risultato[j]) * base;
					supportoParteBassa= supporto;
					riportoParteBassa = riporto;

					if(j % 2 == 0) {

						if( ((unsigned int) supportoParteBassa +  (unsigned int) riportoParteBassa) > 255 ) {
							overflowPari=overflowPari +1;
						}
						//risultatoInt[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
						//risultato[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
						vettoriSupporto[(t %4)*n + j] = (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
						//if( ((unsigned int) risultato[j] +  (unsigned int) riporto16Pari) > 255 ) {
						if( ((unsigned int) vettoriSupporto[(t %4)*n + j] +  (unsigned int) riporto16Pari) > 255 ) {
							overflowPari=overflowPari +1;
						}
						// risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) riporto16Pari );
						vettoriSupporto[(t %4)*n + j] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %4)*n + j] + (unsigned int) riporto16Pari );

						if( ((unsigned int) vettoriSupporto[(t %4)*n + j] +  (unsigned int) overflowDispari) > 255 ) {
							overflowPari=overflowPari +1;
						}
						// risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) overflowDispari );
						vettoriSupporto[(t %4)*n + j] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %4)*n + j] + (unsigned int) overflowDispari );
						overflowDispari=0;
						// risultato[j]= (char) ( (unsigned int) supporto + (unsigned int) riporto + (unsigned int) riporto16Pari);
					} else {

						if( ((unsigned int) supportoParteBassa +  (unsigned int) riportoParteBassa) > 255 ) {
							overflowDispari=overflowDispari +1;
						}
						// risultato[j]= (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );
						vettoriSupporto[(t %4)*n + j] = (unsigned char) ( (unsigned int) supportoParteBassa + (unsigned int) riportoParteBassa );

						if( ((unsigned int) vettoriSupporto[(t %4)*n + j] +  (unsigned int) riporto16Dispari) > 255 ) {
							overflowDispari=overflowDispari +1;
						}
						// risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) riporto16Dispari );
						vettoriSupporto[(t %4)*n + j] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %4)*n + j] + (unsigned int) riporto16Dispari );
						if( ((unsigned int) vettoriSupporto[(t %4)*n + j] +  (unsigned int) overflowPari) > 255 ) {
							overflowDispari=overflowDispari +1;
						}
						//risultato[j]= (unsigned char) ( (unsigned int) risultato[j] + (unsigned int) overflowPari );
						vettoriSupporto[(t %4)*n + j] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %4)*n + j] + (unsigned int) overflowPari );
						overflowPari=0;

						//risultato[j]= (char) ( (unsigned int) supporto + (unsigned int) riporto + (unsigned int) riporto16PDispari);
					}
					supporto= supporto >> 8;

					// supporto = supporto + overflow;
					// overflow=0;
					riporto = supporto;
					if(j % 2 == 0) {
						riporto16Pari = riporto >> 8;
					} else {
						riporto16Dispari = riporto >> 8;

					}
				} // chiusura for
				overflowPari=0;
				riporto16Dispari=0;
				// printf("riporto16Pari = %x \n",riporto16Pari);
				//printf("supporto = %x \n",supporto);
				//printf("overflowDispari = %x \n",overflowDispari);
				if( ((unsigned char) supporto  + (unsigned int) riporto16Pari) > 255 ) {
					overflowPari=1;
					//   printf("overflow viene aumentato 1 \n");
				}
				//risultato[2 + 2*i] = (unsigned char) supporto  + (unsigned int) riporto16Pari ;
				vettoriSupporto[(t %4)*n + 8 + 16*(t-sottraggo1)] =  (unsigned char) supporto  + (unsigned int) riporto16Pari ;

				if( (unsigned int) vettoriSupporto[(t %4)*n + 8 + 16*(t-sottraggo1)]  + (unsigned int) overflowDispari > 255 ) {
					overflowPari = overflowPari+ 1;
					// printf("overflow viene aumentato 2 \n");
				}
				//risultato[2 + 2*i] = (unsigned char) ( (unsigned int) risultato[2 + 2*i] + (unsigned int) overflowDispari );
				vettoriSupporto[(t %4)*n+ 8+ 16*(t-sottraggo1)] = (unsigned char) ( (unsigned int) vettoriSupporto[(t %4)*n+ 8 + 16*(t-sottraggo1)]  + (unsigned int) overflowDispari );

				supporto= supporto >> 8;
				//risultato[2 + 2*i +1] = (unsigned char) supporto  + (unsigned int) overflowPari ;
				vettoriSupporto[(t %4)*n+ 8 + 16*(t-sottraggo1) +1] = (unsigned char) supporto  + (unsigned int) overflowPari ;

				// }

				__syncthreads();

				// i thread pari sommano i due risultati per ora in risultato, riciclo overflow pari e overflow dispari
				overflowPari = 0;
				overflowDispari=0;
				if(t % 4 == 0 || t %4 ==2) { // ATTENZIONE, sostsituire risultato con risultatoTOT

					offsetT1= t*2;
					offsetT2= offsetT1 +2;
					for(k=0; k< 16; k++) {

						risultato[(t-sottraggo1)*16]= vettoriSupporto[t*8];
					}
					risultato[t*8] = vettoriSupporto[t*8];
					risultato[t*8 +1] = vettoriSupporto[t*8 +1];
					if(vettoriSupporto[t*8 +2] + vettoriSupporto[n+ t*8] > 255) {
						overflowPari = 1;
					}
					risultato[t*8 +2] = vettoriSupporto[t*8 +2] + vettoriSupporto[n+ t*8];
					if(vettoriSupporto[t*8 +3] + vettoriSupporto[n+ t*8+1 ] + overflowPari > 255) {
						overflowDispari = 1;
					}
					risultato[t*8 +3] = vettoriSupporto[t*8 +3] + vettoriSupporto[n + t*8 +1 ] +overflowPari;
					if(vettoriSupporto[t*8 +4] + vettoriSupporto[n + t*8 +2 ] + overflowDispari  > 255) {
						overflowPari = 1;
					}
					risultato[t*8 +4] = vettoriSupporto[t*8 +4] + vettoriSupporto[n+ t*8 +2]+ overflowDispari;
					if(vettoriSupporto[t*8 +5] + vettoriSupporto[n + t*8+3 ] + overflowPari > 255) {
						overflowDispari = 1;
					}
					risultato[t*8 +5] = vettoriSupporto[t*8 +5] + vettoriSupporto[n + t*8 +3 ] +overflowPari;
					if(vettoriSupporto[t*8 +6] + vettoriSupporto[n + t*8 +4 ] + overflowDispari  > 255) {
						overflowPari = 1;
					}
					risultato[t*8 +6] = vettoriSupporto[t*8 +6] + vettoriSupporto[n + t*8 +4]+ overflowDispari;
					if(vettoriSupporto[t*8 +7] + vettoriSupporto[n + t*8+5 ] + overflowPari > 255) {
						overflowDispari = 1;
					}
					risultato[t*8 +5] = vettoriSupporto[t*8 +7] + vettoriSupporto[n + t*8 +5 ] +overflowPari;
				}
				__syncthreads();
				if(t== 0) {

					for(k=0; k < 7; k++ ) {

						printf(" hghj risultato[%d] = %x \n",k,risultato[k] );
						printf("ghv risultato[%d] = %x \n",k+1024,risultato[k+1024] );
					}


				}



			} // fine I=2

		}
	   }

return;
}
__global__ void provacuda(void){


	int tid= blockIdx.x*blockDim.x+threadIdx.x;

	if(tid == 0 ) {

		printf("ciao2 \n");

	}
	printf("ciao \n");
}

//int powerBigNumbersGPU(unsigned char *risultato,unsigned int esponente,unsigned int base);
int main(void)
{

   // unsigned int esponente; // numero massimo su 16 bit, deve essere almeno 2
    // unsigned int base;  // numero massimo su 16 bit
     int n =131070;
     unsigned char *dev_vect;
     unsigned char *vect;
     int *dev_variabili;
     int *variabili;
     int i;

     variabili = (int*) malloc(n*9*sizeof(int *));  // in realta variabili è una matrice srotolata

     hipMalloc((void**)&dev_variabili,9*n*sizeof(int));
     hipMemcpy(dev_variabili,variabili,9*n*sizeof(int),hipMemcpyHostToDevice);

     vect = (unsigned char*) malloc(n*sizeof(unsigned char));
     hipMalloc((void**)&dev_vect,n*sizeof(unsigned char));
     hipMemcpy(dev_vect,vect,n*sizeof(unsigned char),hipMemcpyHostToDevice);

	// il numero massimo di moltiplicazioni è 65535, ogni thread gestisce 2 moltiplicazioni
	// quindi il numero di thread necessari è 32
     provacuda<<<1,10>>>();
	//memoriaGlobale<<<64,1024>>>(dev_vect, n,numeroBlocchi);
	powerBigNumbersGPUShared<<<32,1024>>>(dev_vect,dev_variabili, 4096,7);


	hipDeviceReset();
		  return 0;
	// 7^15 = 45160B7A437
	/*
  const int n = 1048576;
  int numeroBlocchi = 10;
  int *giapassati ;
  int *giapassatihost;
  int i;
  //double  a[n], r[n], d[n];


  double *vect;
  double *dev_vect;

  double *vect2;
  double *dev_vect2;

  vect = (double*) malloc(n*sizeof(double));
  hipMalloc((void**)&dev_vect,n*sizeof(double));
  hipMemcpy(dev_vect,vect,n*sizeof(double),hipMemcpyHostToDevice);
 // hipMalloc(&d_d, n * sizeof(double));
  // run version with static shared memory
  //hipMemcpy(d_d, a, n*sizeof(double), hipMemcpyHostToDevice);


  memoriaGlobale<<<64,1024>>>(dev_vect, n,numeroBlocchi);
  hipMemcpy(vect, dev_vect, n*sizeof(double), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++) {
	  if(vect[i] != i) {

		  printf("errore 1 ! \n" );
		  printf("d[%i] = %f \n",i,vect[i]);
	  }
	  if(i== 0) {
	      printf("d[%i] = %f \n",i,vect[i]);
	  	  }
	  if(i== 1048575) {
    printf("d[%i] = %f \n",i,vect[i]);
	  }
  }



  hipMalloc((void**)&giapassati,384*sizeof(int));
  vect2 = (double*) malloc(n*sizeof(double));
  giapassatihost = (int*) malloc(384*sizeof(int));
  for(i=0; i< 384 ; i++) {

    	  		giapassatihost[i] = 0;

    	  	}
  hipMemcpy(giapassati,giapassatihost,384*sizeof(int),hipMemcpyHostToDevice);
   hipMalloc((void**)&dev_vect2,n*sizeof(double));
   hipMemcpy(dev_vect2,vect2,n*sizeof(double),hipMemcpyHostToDevice);
  // hipMalloc(&d_d, n * sizeof(double));
   // run version with static shared memory
   //hipMemcpy(d_d, a, n*sizeof(double), hipMemcpyHostToDevice);

   for(i=0; i< 384 ; i++) {

  	  		//giapassati[i] = 0;

  	  	}

   memoriaShared<<<1,385>>>(dev_vect2, giapassati);
   hipMemcpy(vect2, dev_vect2, n*sizeof(double), hipMemcpyDeviceToHost);
   for (int i = 0; i < n; i++) {
	   if(vect2[i] != i) {

		   printf("errore 2 ! \n" );
		   printf("d[%i] = %f \n",i,vect2[i]);
	   }
	  // printf("d[%i] = %f \n",i,vect2[i]);
 	  if(i== 1048575) {
     printf("d[%i] = %f \n",i,vect2[i]);
 	  }
   }
  // run dynamic shared memory version
 // hipMemcpy(d_d, a, n*sizeof(int), hipMemcpyHostToDevice);
 // dynamicReverse<<<1,n,n*sizeof(int)>>>(d_d, n);
 // hipMemcpy(d, d_d, n * sizeof(int), hipMemcpyDeviceToHost);
 // for (int i = 0; i < n; i++)
  //  if (d[i] != r[i]) printf("Error: d[%d]!=r[%d] (%d, %d)n", i, i, d[i], r[i]);
  */

}





